#include "hip/hip_runtime.h"
#include "cuda/kdtree.cuh"

namespace cuda
{
   __host__ __device__
   inline int divideUp(int a, int b)
   {
      return (a + b - 1) / b;
   }

   __host__ __device__
   int getSampleNum(int x)
   {
      return divideUp( x, SampleStride );
   }

   __device__
   static inline int getNextPowerOfTwo(int x)
   {
      /*
         --x;
         x |= x >> 1;
         x |= x >> 2;
         x |= x >> 4;
         x |= x >> 8;
         x |= x >> 16;
         return ++x;
       */
      constexpr int bits = sizeof( int ) * 8;
      return 1 << (bits - __clz( x - 1 ));
   }

   KdtreeCUDA::KdtreeCUDA(const node_type* vertices, int size, int dim) : Dim( dim ), NodeNum( 0 )
   {
      if (DeviceNum == 0) prepareCUDA();
      create( vertices, size );
   }

   void KdtreeCUDA::prepareCUDA()
   {
      CHECK_CUDA( hipGetDeviceCount( &DeviceNum ) );
      DeviceNum = std::min( DeviceNum, 2 );

      int gpu_num = 0;
      std::array<int, 2> gpu_id{};
      hipDeviceProp_t properties[2];
      for (int i = 0; i < DeviceNum; ++i) {
         CHECK_CUDA( hipGetDeviceProperties( &properties[i], i ) );
         if (isP2PCapable( properties[i] )) gpu_id[gpu_num++] = i;
      }

      if (gpu_num == 2) {
         int can_access_peer_01, can_access_peer_10;
		   CHECK_CUDA( hipDeviceCanAccessPeer( &can_access_peer_01, gpu_id[0], gpu_id[1] ) );
		   CHECK_CUDA( hipDeviceCanAccessPeer( &can_access_peer_10, gpu_id[1], gpu_id[0] ) );
         if (can_access_peer_01 == 0 || can_access_peer_10 == 0) {
            CHECK_CUDA( hipSetDevice( gpu_id[0] ) );
            DeviceNum = 1;
         }
         else {
            CHECK_CUDA( hipSetDevice( gpu_id[0] ) );
            CHECK_CUDA( hipDeviceEnablePeerAccess( gpu_id[1], 0 ) );
            CHECK_CUDA( hipSetDevice( gpu_id[1] ) );
            CHECK_CUDA( hipDeviceEnablePeerAccess( gpu_id[0], 0 ) );

            const bool has_uva = properties[gpu_id[0]].unifiedAddressing && properties[gpu_id[1]].unifiedAddressing;
            if (!has_uva) DeviceNum = 1;
         }
      }
      else DeviceNum = 1;

      Sort.resize( DeviceNum );
      Root.resize( DeviceNum );
      Streams.resize( DeviceNum );
      SyncEvents.resize( DeviceNum );
      References.resize( DeviceNum );
      Buffers.resize( DeviceNum );
      CoordinatesDevicePtr.resize( DeviceNum );
      for (int i = 0; i < DeviceNum; ++i) {
         DeviceID.emplace_back( i );

         setDevice( i );
         CHECK_CUDA( hipStreamCreate( &Streams[i] ) );
         CHECK_CUDA( hipEventCreate( &SyncEvents[i] ) );
      }
   }

   __global__
   void cuInitialize(KdtreeNode* root, int size)
   {
      auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      auto step = static_cast<int>(blockDim.x * gridDim.x);
      for (int i = index; i < size; i += step) {
         root[i].Index = i;
         root[i].LeftChildIndex = -1;
         root[i].RightChildIndex = -1;
      }
   }

   void KdtreeCUDA::initialize(const node_type* coordinates, int size, int device_id)
   {
      if (CoordinatesDevicePtr[device_id] != nullptr) {
         throw std::runtime_error( "coordinates device ptr already allocated!" );
      }
      if (Root[device_id] != nullptr) throw std::runtime_error( "k-d tree already allocated!" );

      setDevice( device_id );
      CHECK_CUDA(
         hipMalloc(
            reinterpret_cast<void**>(&CoordinatesDevicePtr[device_id]),
            sizeof( node_type ) * Dim * (size + 1)
         )
      );
      CHECK_CUDA(
         hipMemcpyAsync(
            CoordinatesDevicePtr[device_id], coordinates, sizeof( node_type ) * Dim * size,
            hipMemcpyHostToDevice, Streams[device_id]
         )
      );

      node_type max_value[Dim];
      for (int i = 0; i < Dim; ++i) max_value[i] = std::numeric_limits<node_type>::max();
      CHECK_CUDA(
         hipMemcpyAsync(
            CoordinatesDevicePtr[device_id] + size * Dim, max_value, sizeof( node_type ) * Dim,
            hipMemcpyHostToDevice, Streams[device_id]
         )
      );

      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Root[device_id]), sizeof( KdtreeNode ) * size ) );

      cuInitialize<<<ThreadBlockNum, ThreadNum, 0, Streams[device_id]>>>( Root[device_id], size );
   }

   __global__
   void cuInitializeReference(int* reference, int size)
   {
      auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      auto step = static_cast<int>(blockDim.x * gridDim.x);
      for (int i = index; i < size; i += step) {
         reference[i] = i;
      }
   }

   void KdtreeCUDA::initializeReference(int size, int axis, int device_id)
   {
      setDevice( device_id );
      int** references = References[device_id];
      for (int i = 0; i <= Dim + 1; ++i) {
         if (references[i] == nullptr) {
            CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&references[i]), sizeof( int ) * size ) );
         }
      }
      cuInitializeReference<<<ThreadBlockNum, ThreadNum, 0, Streams[device_id]>>>( references[axis], size );
   }

   __global__
   void cuCopyCoordinates(
      node_type* target,
      const node_type* coordinates,
      const int* reference,
      int size,
      int axis,
      int dim
   )
   {
      auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      auto step = static_cast<int>(blockDim.x * gridDim.x);
      for (int i = index; i < size; i += step) {
         target[i] = coordinates[reference[i] * dim + axis];
      }
   }

   __device__
   node_type compareSuperKey(
      node_type front_a,
      node_type front_b,
      const node_type* a,
      const node_type* b,
      int axis,
      int dim
   )
   {
      node_type difference = front_a - front_b;
      for (int i = 1; difference == 0 && i < dim; ++i) {
         int r = i + axis;
         r = r < dim ? r : r - dim;
         difference = a[r] - b[r];
      }
      return difference;
   }

   __device__
   int searchInclusively(
      int index,
      node_type value,
      const int* reference,
      const node_type* buffer,
      const node_type* coordinates,
      int length,
      int step,
      int axis,
      int dim
   )
   {
      if (length == 0) return 0;

      int i = 0;
      while (step > 0) {
         const int j = min( i + step, length );
         const node_type t = compareSuperKey(
            buffer[j - 1], value, coordinates + reference[j - 1] * dim, coordinates + index * dim, axis, dim
         );
         if (t <= 0) i = j;
         step >>= 1;
      }
      return i;
   }

   __device__
   int searchExclusively(
      int index,
      node_type value,
      const int* reference,
      const node_type* buffer,
      const node_type* coordinates,
      int length,
      int step,
      int axis,
      int dim
   )
   {
      if (length == 0) return 0;

      int i = 0;
      while (step > 0) {
         const int j = min( i + step, length );
         const node_type t = compareSuperKey(
            buffer[j - 1], value, coordinates + reference[j - 1] * dim, coordinates + index * dim, axis, dim
         );
         if (t < 0) i = j;
         step >>= 1;
      }
      return i;
   }

   __global__
   void cuSort(
      int* target_reference,
      node_type* target_buffer,
      const int* source_reference,
      const node_type* source_buffer,
      const node_type* coordinates,
      int axis,
      int dim
   )
   {
      __shared__ int reference[SharedSizeLimit];
      __shared__ node_type buffer[SharedSizeLimit];

      source_buffer += blockIdx.x * SharedSizeLimit + threadIdx.x;
      source_reference += blockIdx.x * SharedSizeLimit + threadIdx.x;
      target_buffer += blockIdx.x * SharedSizeLimit + threadIdx.x;
      target_reference += blockIdx.x * SharedSizeLimit + threadIdx.x;
      buffer[threadIdx.x] = source_buffer[0];
      reference[threadIdx.x] = source_reference[0];
      buffer[threadIdx.x + SharedSizeLimit / 2] = source_buffer[SharedSizeLimit / 2];
      reference[threadIdx.x + SharedSizeLimit / 2] = source_reference[SharedSizeLimit / 2];

      for (int step = 1; step < SharedSizeLimit; step <<= 1) {
         const int i = static_cast<int>(threadIdx.x) & (step - 1);
         node_type* base_buffer = buffer + 2 * (threadIdx.x - i);
         int* base_reference = reference + 2 * (threadIdx.x - i);

         __syncthreads();
         const node_type buffer_x = base_buffer[i];
         const int reference_x = base_reference[i];
         const node_type buffer_y = base_buffer[i + step];
         const int reference_y = base_reference[i + step];
         const int x = searchExclusively(
            reference_x, buffer_x, base_reference + step, base_buffer + step, coordinates, step, step, axis, dim
         ) + i;
         const int y = searchExclusively(
            reference_y, buffer_y, base_reference, base_buffer, coordinates, step, step, axis, dim
         ) + i;

         __syncthreads();
         base_buffer[x] = buffer_x;
         base_buffer[y] = buffer_y;
         base_reference[x] = reference_x;
         base_reference[y] = reference_y;
      }

      __syncthreads();
      target_buffer[0] = buffer[threadIdx.x];
      target_reference[0] = reference[threadIdx.x];
      target_buffer[SharedSizeLimit / 2] = buffer[threadIdx.x + SharedSizeLimit / 2];
      target_reference[SharedSizeLimit / 2] = reference[threadIdx.x + SharedSizeLimit / 2];
   }

   __global__
   void cuGenerateSampleRanks(
      int* ranks_a,
      int* ranks_b,
      int* reference,
      node_type* buffer,
      const node_type* coordinates,
      int step,
      int size,
      int axis,
      int dim,
      int thread_num
   )
   {
      auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      if (index >= thread_num) return;

      const int i = index % (step / SampleStride - 1);
      const int segment_base = (index - i) * 2 * SampleStride;
      buffer += segment_base;
      reference += segment_base;
      ranks_a += segment_base / SampleStride;
      ranks_b += segment_base / SampleStride;

      const int element_a = step;
      const int element_b = min( step, size - step - segment_base );
      const int sample_a = getSampleNum( element_a );
      const int sample_b = getSampleNum( element_b );
      if (i < sample_a) {
         ranks_a[i] = i * SampleStride;
         ranks_b[i] = searchExclusively(
            reference[i * SampleStride], buffer[i * SampleStride],
            reference + step, buffer + step, coordinates,
            element_b, getNextPowerOfTwo( element_b ), axis, dim
         );
      }
      if (i < sample_b) {
         ranks_b[step / SampleStride + i] = i * SampleStride;
         ranks_a[step / SampleStride + i] = searchInclusively(
            reference[i * SampleStride + step], buffer[i * SampleStride + step],
            reference, buffer, coordinates,
            element_a, getNextPowerOfTwo( element_a ), axis, dim
         );
      }
   }

   __global__
   void cuMergeRanksAndIndices(int* limits, const int* ranks, int step, int size, int thread_num)
   {
      auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      if (index >= thread_num) return;

      const int i = index % (step / SampleStride - 1);
      const int segment_base = (index - i) * 2 * SampleStride;
      ranks += (index - i) * 2;
      limits += (index - i) * 2;

      const int element_a = step;
      const int element_b = min( step, size - step - segment_base );
      const int sample_a = getSampleNum( element_a );
      const int sample_b = getSampleNum( element_b );
      if (i < sample_a) {
         int x = 0;
         if (sample_b != 0) {
            const int stride = getNextPowerOfTwo( sample_b );
            while (step > 0) {
               const int j = min( x + stride, sample_b );
               if (ranks[sample_a + j - 1] < ranks[i]) x = j;
               step >>= 1;
            }
         }
         limits[x + i] = ranks[i];
      }
      if (i < sample_b) {
         int x = 0;
         if (sample_a != 0) {
            const int stride = getNextPowerOfTwo( sample_a );
            while (step > 0) {
               const int j = min( x + stride, sample_a );
               if (ranks[j - 1] <= ranks[sample_a + i]) x = j;
               step >>= 1;
            }
         }
         limits[x + i] = ranks[sample_a + i];
      }
   }

   __device__
   void merge(
      int* reference,
      node_type* buffer,
      const node_type* coordinates,
      int length_a,
      int length_b,
      int axis,
      int dim
   )
   {
      const int* reference_a = reference;
      const int* reference_b = reference + SampleStride;
      const node_type* buffer_a = buffer;
      const node_type* buffer_b = buffer + SampleStride;

      int index_a, index_b, x, y;
      node_type value_a, value_b;
      if (threadIdx.x < length_a) {
         value_a = buffer_a[threadIdx.x];
         index_a = reference_a[threadIdx.x];
         x = static_cast<int>(threadIdx.x) +
            searchExclusively( index_a, value_a, reference_b, buffer_b, coordinates, length_b, SampleStride, axis, dim );
      }
      if (threadIdx.x < length_b) {
         value_b = buffer_b[threadIdx.x];
         index_b = reference_b[threadIdx.x];
         y = static_cast<int>(threadIdx.x) +
            searchInclusively( index_b, value_b, reference_a, buffer_a, coordinates, length_a, SampleStride, axis, dim );
      }

      __syncthreads();
      if (threadIdx.x < length_a) {
         buffer[x] = value_a;
         reference[x] = index_a;
      }
      if (threadIdx.x < length_b) {
         buffer[y] = value_b;
         reference[y] = index_b;
      }
   }

   __global__
   void cuMergeReferences(
      int* target_reference,
      node_type* target_buffer,
      const int* source_reference,
      const node_type* source_buffer,
      const node_type* coordinates,
      const int* limits_a,
      const int* limits_b,
      int step,
      int size,
      int axis,
      int dim
   )
   {
      __shared__ int reference[2 * SampleStride];
      __shared__ node_type buffer[2 * SampleStride];

      const int index = static_cast<int>(blockIdx.x) & (2 * step / SampleStride - 1);
      const int segment_base = (static_cast<int>(blockIdx.x) - index) * SampleStride;
      target_buffer += segment_base;
      target_reference += segment_base;
      source_buffer += segment_base;
      source_reference += segment_base;

      __shared__ int start_source_a, start_source_b;
      __shared__ int start_target_a, start_target_b;
      __shared__ int length_a, length_b;

      if (threadIdx.x == 0) {
         const int element_a = step;
         const int element_b = min( step, size - step - segment_base );
         const int sample_a = getSampleNum( element_a );
         const int sample_b = getSampleNum( element_b );
         const int sample_num = sample_a + sample_b;
         start_source_a = limits_a[blockIdx.x];
         start_source_b = limits_b[blockIdx.x];
         const int end_source_a = index + 1 < sample_num ? limits_a[blockIdx.x + 1] : element_a;
         const int end_source_b = index + 1 < sample_num ? limits_b[blockIdx.x + 1] : element_b;
         length_a = end_source_a - start_source_a;
         length_b = end_source_b - start_source_b;
         start_target_a = start_source_a + start_source_b;
         start_target_b = start_target_a + length_a;
      }

      __syncthreads();
      if (threadIdx.x < length_a) {
         buffer[threadIdx.x] = source_buffer[start_source_a + threadIdx.x];
         reference[threadIdx.x] = source_reference[start_source_a + threadIdx.x];
      }
      if (threadIdx.x < length_b) {
         buffer[threadIdx.x + SampleStride] = source_buffer[start_source_b + threadIdx.x + step];
         reference[threadIdx.x + SampleStride] = source_reference[start_source_b + threadIdx.x + step];
      }

      __syncthreads();
      merge( reference, buffer, coordinates, length_a, length_b, axis, dim );

      __syncthreads();
      if (threadIdx.x < length_a) {
         target_buffer[start_target_a + threadIdx.x] = buffer[threadIdx.x];
         target_reference[start_target_a + threadIdx.x] = reference[threadIdx.x];
      }
      if (threadIdx.x < length_b) {
         target_buffer[start_target_b + threadIdx.x] = buffer[length_a + threadIdx.x];
         target_reference[start_target_b + threadIdx.x] = reference[length_a + threadIdx.x];
      }
   }

   void KdtreeCUDA::sortPartially(
      int source_index,
      int target_index,
      int start_offset,
      int size,
      int axis,
      int device_id
   )
   {
      assert( CoordinatesDevicePtr[device_id] != nullptr );
      assert( References[device_id][source_index] != nullptr && References[device_id][target_index] != nullptr );

      setDevice( device_id );
      if (Buffers[device_id][source_index] == nullptr) {
         CHECK_CUDA(
            hipMalloc( reinterpret_cast<void**>(&Buffers[device_id][source_index]), sizeof( node_type ) * size )
         );
         cuCopyCoordinates<<<ThreadBlockNum, ThreadNum, 0, Streams[device_id]>>>(
            Buffers[device_id][source_index],
            CoordinatesDevicePtr[device_id] + start_offset * Dim,
            References[device_id][source_index],
            size,
            axis,
            Dim
         );
      }
      if (Buffers[device_id][target_index] == nullptr) {
         CHECK_CUDA(
            hipMalloc( reinterpret_cast<void**>(&Buffers[device_id][target_index]), sizeof( node_type ) * size )
         );
      }

      int stage_num = 0;
      for (int step = SharedSizeLimit; step < size; step <<= 1) stage_num++;

      int* in_reference = nullptr;
      int* out_reference = nullptr;
      node_type* in_buffer = nullptr;
      node_type* out_buffer = nullptr;
      if (stage_num & 1) {
         in_buffer = Sort[device_id].Buffer;
         in_reference = Sort[device_id].Reference;
         out_buffer = Buffers[device_id][target_index];
         out_reference = References[device_id][target_index] + start_offset;
      }
      else {
         in_buffer = Buffers[device_id][target_index];
         in_reference = References[device_id][target_index] + start_offset;
         out_buffer = Sort[device_id].Buffer;
         out_reference = Sort[device_id].Reference;
      }

      assert( size <= SampleStride * Sort[device_id].MaxSampleNum );
      assert( size % SharedSizeLimit == 0 );

      const int block_num = size / SharedSizeLimit;
      int thread_num = SharedSizeLimit / 2;
      cuSort<<<block_num, thread_num, 0, Streams[device_id]>>>(
         in_reference, in_buffer,
         References[device_id][source_index] + start_offset, Buffers[device_id][source_index],
         CoordinatesDevicePtr[device_id], axis, Dim
      );

      for (int step = SharedSizeLimit; step < size; step <<= 1) {
         const int last = size % (2 * step);
         thread_num = last > step ? (size + 2 * step - last) / (2 * SampleStride) : (size - last) / (2 * SampleStride);
         cuGenerateSampleRanks<<<divideUp( thread_num, 256 ), 256, 0, Streams[device_id]>>>(
            Sort[device_id].RanksA, Sort[device_id].RanksB,
            in_reference, in_buffer, CoordinatesDevicePtr[device_id],
            step, size, axis, Dim, thread_num
         );
         cuMergeRanksAndIndices<<<divideUp( thread_num, 256 ), 256, 0, Streams[device_id]>>>(
            Sort[device_id].LimitsA, Sort[device_id].RanksA, step, size, thread_num
         );
         cuMergeRanksAndIndices<<<divideUp( thread_num, 256 ), 256, 0, Streams[device_id]>>>(
            Sort[device_id].LimitsB, Sort[device_id].RanksB, step, size, thread_num
         );

         const int merge_pairs = last > step ? getSampleNum( size ) : (size - last) / SampleStride;
         cuMergeReferences<<<merge_pairs, SampleStride, 0, Streams[device_id]>>>(
            out_reference, out_buffer,
            in_reference, in_buffer, CoordinatesDevicePtr[device_id],
            Sort[device_id].LimitsA, Sort[device_id].LimitsB,
            step, size, axis, Dim
         );
      }
   }

   void KdtreeCUDA::sort(int* end, int size)
   {
      const int max_sample_num = size / SampleStride + 1;
      for (int i = 0; i < DeviceNum; ++i) {
         setDevice( i );
         Sort[i].MaxSampleNum = max_sample_num;
         CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Sort[i].RanksA), sizeof( int ) * max_sample_num ) );
         CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Sort[i].RanksB), sizeof( int ) * max_sample_num ) );
         CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Sort[i].LimitsA), sizeof( int ) * max_sample_num ) );
         CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Sort[i].LimitsB), sizeof( int ) * max_sample_num ) );
         CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Sort[i].Reference), sizeof( int ) * size ) );
         CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Sort[i].Buffer), sizeof( node_type ) * size ) );
      }

      const int size_per_device = size / DeviceNum;
      if (DeviceNum > 1) {
         for (int i = 0; i < DeviceNum; ++i) {
            initializeReference( size_per_device, 0, i );
            sortPartially( 0, Dim, 0, size_per_device, 0, i );
         }
      }
      else {

      }
      sync();
   }

   void KdtreeCUDA::create(const node_type* coordinates, int size)
   {
      const int size_per_device = size / DeviceNum;
      for (int i = 0; i < DeviceNum; ++i) {
         const node_type* ptr = coordinates + i * Dim * size_per_device;
         initialize( ptr, size_per_device, i );
      }
      hipDeviceSynchronize();

      int end[Dim];
      sort( end, size );
   }
}