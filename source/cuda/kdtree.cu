#include "hip/hip_runtime.h"
#include "cuda/kdtree.cuh"

namespace cuda
{
   KdtreeCUDA::KdtreeCUDA(const node_type* vertices, int size, int dim) : Dim( dim ), NodeNum( 0 )
   {
      if (DeviceNum == 0) prepareCUDA();
      create( vertices, size );
   }

   void KdtreeCUDA::prepareCUDA()
   {
      CHECK_CUDA( hipGetDeviceCount( &DeviceNum ) );
      DeviceNum = std::min( DeviceNum, 2 );

      int gpu_num = 0;
      std::array<int, 2> gpu_id{};
      hipDeviceProp_t properties[2];
      for (int i = 0; i < DeviceNum; ++i) {
         CHECK_CUDA( hipGetDeviceProperties( &properties[i], i ) );
         if (isP2PCapable( properties[i] )) gpu_id[gpu_num++] = i;
      }

      if (gpu_num == 2) {
         int can_access_peer_01, can_access_peer_10;
		   CHECK_CUDA( hipDeviceCanAccessPeer( &can_access_peer_01, gpu_id[0], gpu_id[1] ) );
		   CHECK_CUDA( hipDeviceCanAccessPeer( &can_access_peer_10, gpu_id[1], gpu_id[0] ) );
         if (can_access_peer_01 == 0 || can_access_peer_10 == 0) {
            CHECK_CUDA( hipSetDevice( gpu_id[0] ) );
            DeviceNum = 1;
         }
         else {
            CHECK_CUDA( hipSetDevice( gpu_id[0] ) );
            CHECK_CUDA( hipDeviceEnablePeerAccess( gpu_id[1], 0 ) );
            CHECK_CUDA( hipSetDevice( gpu_id[1] ) );
            CHECK_CUDA( hipDeviceEnablePeerAccess( gpu_id[0], 0 ) );

            const bool has_uva = properties[gpu_id[0]].unifiedAddressing && properties[gpu_id[1]].unifiedAddressing;
            if (!has_uva) DeviceNum = 1;
         }
      }
      else DeviceNum = 1;

      Sort.resize( DeviceNum );
      Root.resize( DeviceNum );
      Streams.resize( DeviceNum );
      SyncEvents.resize( DeviceNum );
      References.resize( DeviceNum );
      Buffers.resize( DeviceNum );
      CoordinatesDevicePtr.resize( DeviceNum );
      for (int i = 0; i < DeviceNum; ++i) {
         DeviceID.emplace_back( i );

         setDevice( i );
         CHECK_CUDA( hipStreamCreate( &Streams[i] ) );
         CHECK_CUDA( hipEventCreate( &SyncEvents[i] ) );
      }
   }

   __global__
   void cuInitialize(KdtreeNode* root, int size)
   {
      auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      auto step = static_cast<int>(blockDim.x * gridDim.x);
      for (int i = index; i < size; i += step) {
         root[i].Index = i;
         root[i].LeftChildIndex = -1;
         root[i].RightChildIndex = -1;
      }
   }

   void KdtreeCUDA::initialize(const node_type* coordinates, int size, int device_id)
   {
      if (CoordinatesDevicePtr[device_id] != nullptr) {
         throw std::runtime_error( "coordinates device ptr already allocated!" );
      }
      if (Root[device_id] != nullptr) throw std::runtime_error( "k-d tree already allocated!" );

      setDevice( device_id );
      CHECK_CUDA(
         hipMalloc(
            reinterpret_cast<void**>(&CoordinatesDevicePtr[device_id]),
            sizeof( node_type ) * Dim * (size + 1)
         )
      );
      CHECK_CUDA(
         hipMemcpyAsync(
            CoordinatesDevicePtr[device_id], coordinates, sizeof( node_type ) * Dim * size,
            hipMemcpyHostToDevice, Streams[device_id]
         )
      );

      node_type max_value[Dim];
      for (int i = 0; i < Dim; ++i) max_value[i] = std::numeric_limits<node_type>::max();
      CHECK_CUDA(
         hipMemcpyAsync(
            CoordinatesDevicePtr[device_id] + size * Dim, max_value, sizeof( node_type ) * Dim,
            hipMemcpyHostToDevice, Streams[device_id]
         )
      );

      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Root[device_id]), sizeof( KdtreeNode ) * size ) );

      cuInitialize<<<ThreadBlockNum, ThreadNum, 0, Streams[device_id]>>>( Root[device_id], size );
   }

   __global__
   void cuInitializeReference(int* reference, int size)
   {
      auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      auto step = static_cast<int>(blockDim.x * gridDim.x);
      for (int i = index; i < size; i += step) {
         reference[i] = i;
      }
   }

   void KdtreeCUDA::initializeReference(int size, int axis, int device_id)
   {
      setDevice( device_id );
      int** references = References[device_id];
      for (int i = 0; i <= Dim + 1; ++i) {
         if (references[i] == nullptr) {
            CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&references[i]), sizeof( int ) * size ) );
         }
      }
      cuInitializeReference<<<ThreadBlockNum, ThreadNum, 0, Streams[device_id]>>>( references[axis], size );
   }

   __global__
   void cuCopyCoordinates(
      node_type* target,
      const node_type* coordinates,
      const int* reference,
      int size,
      int axis,
      int dim
   )
   {
      auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      auto step = static_cast<int>(blockDim.x * gridDim.x);
      for (int i = index; i < size; i += step) {
         target[i] = coordinates[reference[i] * dim + axis];
      }
   }

   __device__
   node_type compareSuperKey(
      node_type front_a,
      node_type front_b,
      const node_type* a,
      const node_type* b,
      int axis,
      int dim
   )
   {
      node_type difference = front_a - front_b;
      for (int i = 1; difference == 0 && i < dim; ++i) {
         int r = i + axis;
         r = r < dim ? r : r - dim;
         difference = a[r] - b[r];
      }
      return difference;
   }

   __device__
   uint searchExclusively(
      const node_type* coordinates,
      const node_type* buffer,
      const int* reference,
      node_type value,
      int index,
      uint length,
      uint step,
      int axis,
      int dim
   )
   {
      if (length == 0) return 0;

      uint i = 0;
      while (step > 0) {
         uint j = umin( i + step, length );
         const node_type t = compareSuperKey(
            buffer[j - 1], value, coordinates + reference[j - 1] * dim, coordinates + index * dim, axis, dim
         );
         if (t < 0) i = j;
         step >>= 1;
      }
      return i;
   }

   __global__
   void cuSort(
      int* target_reference,
      node_type* target_buffer,
      const int* source_reference,
      const node_type* source_buffer,
      const node_type* coordinates,
      int axis,
      int dim
   )
   {
      __shared__ int reference[SharedSizeLimit];
      __shared__ node_type buffer[SharedSizeLimit];

      source_buffer += blockIdx.x * SharedSizeLimit + threadIdx.x;
      source_reference += blockIdx.x * SharedSizeLimit + threadIdx.x;
      target_buffer += blockIdx.x * SharedSizeLimit + threadIdx.x;
      target_reference += blockIdx.x * SharedSizeLimit + threadIdx.x;
      buffer[threadIdx.x] = source_buffer[0];
      reference[threadIdx.x] = source_reference[0];
      buffer[threadIdx.x + SharedSizeLimit / 2] = source_buffer[SharedSizeLimit / 2];
      reference[threadIdx.x + SharedSizeLimit / 2] = source_reference[SharedSizeLimit / 2];

      for (uint step = 1; step < SharedSizeLimit; step <<= 1) {
         const uint i = threadIdx.x & (step - 1);
         node_type* base_buffer = buffer + 2 * (threadIdx.x - i);
         int* base_reference = reference + 2 * (threadIdx.x - i);

         __syncthreads();
         const node_type buffer_x = base_buffer[i];
         const int reference_x = base_reference[i];
         const node_type buffer_y = base_buffer[i + step];
         const int reference_y = base_reference[i + step];
         const uint x = searchExclusively(
            coordinates, base_buffer + step, base_reference + step, buffer_x, reference_x, step, step, axis, dim
         ) + i;
         const uint y = searchExclusively(
            coordinates, base_buffer, base_reference, buffer_y, reference_y, step, step, axis, dim
         ) + i;

         __syncthreads();
         base_buffer[x] = buffer_x;
         base_buffer[y] = buffer_y;
         base_reference[x] = reference_x;
         base_reference[y] = reference_y;
      }

      __syncthreads();
      target_buffer[0] = buffer[threadIdx.x];
      target_reference[0] = reference[threadIdx.x];
      target_buffer[SharedSizeLimit / 2] = buffer[threadIdx.x + SharedSizeLimit / 2];
      target_reference[SharedSizeLimit / 2] = reference[threadIdx.x + SharedSizeLimit / 2];
   }

   void KdtreeCUDA::sortPartially(
      int source_index,
      int target_index,
      int start_offset,
      int size,
      int axis,
      int device_id
   )
   {
      assert( CoordinatesDevicePtr[device_id] != nullptr );
      assert( References[device_id][source_index] != nullptr && References[device_id][target_index] != nullptr );

      setDevice( device_id );
      if (Buffers[device_id][source_index] == nullptr) {
         CHECK_CUDA(
            hipMalloc( reinterpret_cast<void**>(&Buffers[device_id][source_index]), sizeof( node_type ) * size )
         );
         cuCopyCoordinates<<<ThreadBlockNum, ThreadNum, 0, Streams[device_id]>>>(
            Buffers[device_id][source_index],
            CoordinatesDevicePtr[device_id] + start_offset * Dim,
            References[device_id][source_index],
            size,
            axis,
            Dim
         );
      }
      if (Buffers[device_id][target_index] == nullptr) {
         CHECK_CUDA(
            hipMalloc( reinterpret_cast<void**>(&Buffers[device_id][target_index]), sizeof( node_type ) * size )
         );
      }

      uint stage_num = 0;
      for (uint step = SharedSizeLimit; step < size; step <<= 1) stage_num++;

      int* in_reference = nullptr;
      int* out_reference = nullptr;
      node_type* in_buffer = nullptr;
      node_type* out_buffer = nullptr;
      if (stage_num & 1) {
         in_buffer = Sort[device_id].Buffer;
         in_reference = Sort[device_id].Reference;
         out_buffer = Buffers[device_id][target_index];
         out_reference = References[device_id][target_index] + start_offset;
      }
      else {
         in_buffer = Buffers[device_id][target_index];
         in_reference = References[device_id][target_index] + start_offset;
         out_buffer = Sort[device_id].Buffer;
         out_reference = Sort[device_id].Reference;
      }

      assert( size <= SampleStride * Sort[device_id].MaxSampleNum );
      assert( size % SharedSizeLimit == 0 );

      const uint block_num = size / SharedSizeLimit;
      const uint thread_num = SharedSizeLimit / 2;
      cuSort<<<block_num, thread_num, 0, Streams[device_id]>>>(
         in_reference, in_buffer,
         References[device_id][source_index] + start_offset, Buffers[device_id][source_index],
         CoordinatesDevicePtr[device_id], axis, Dim
      );

      for (uint step = SharedSizeLimit; step < size; step <<= 1) {

      }
   }

   void KdtreeCUDA::sort(int* end, int size)
   {
      const int max_sample_num = size / SampleStride + 1;
      for (int i = 0; i < DeviceNum; ++i) {
         setDevice( i );
         Sort[i].MaxSampleNum = max_sample_num;
         CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Sort[i].RanksA), sizeof( uint ) * max_sample_num ) );
         CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Sort[i].RanksB), sizeof( uint ) * max_sample_num ) );
         CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Sort[i].LimitsA), sizeof( uint ) * max_sample_num ) );
         CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Sort[i].LimitsB), sizeof( uint ) * max_sample_num ) );
         CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Sort[i].Reference), sizeof( int ) * size ) );
         CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Sort[i].Buffer), sizeof( node_type ) * size ) );
      }

      const int size_per_device = size / DeviceNum;
      if (DeviceNum > 1) {
         for (int i = 0; i < DeviceNum; ++i) {
            initializeReference( size_per_device, 0, i );

         }
      }
      else {

      }
      sync();
   }

   void KdtreeCUDA::create(const node_type* coordinates, int size)
   {
      const int size_per_device = size / DeviceNum;
      for (int i = 0; i < DeviceNum; ++i) {
         const node_type* ptr = coordinates + i * Dim * size_per_device;
         initialize( ptr, size_per_device, i );
      }
      hipDeviceSynchronize();

      int end[Dim];
      sort( end, size );
   }
}