#include "hip/hip_runtime.h"
#include "cuda/kdtree.cuh"

#ifdef USE_CUDA
namespace cuda
{
   __host__ __device__
   inline int divideUp(int a, int b)
   {
      return (a + b - 1) / b;
   }

   __host__ __device__
   inline int getSampleNum(int x)
   {
      return divideUp( x, SampleStride );
   }

   __device__
   static inline int getNextPowerOfTwo(int x)
   {
      constexpr int bits = sizeof( int ) * 8;
      return 1 << (bits - __clz( x - 1 ));
   }

   KdtreeCUDA::KdtreeCUDA(const node_type* vertices, int size, int dim) :
      Coordinates( vertices ), Dim( dim ), TupleNum( size ), NodeNum( 0 )
   {
      prepareCUDA();
      create();
   }

   KdtreeCUDA::~KdtreeCUDA()
   {
      if (!Device.Reference.empty()) {
         for (int axis = 0; axis <= Dim; ++axis) {
            if (Device.Reference[axis] != nullptr) hipFree( Device.Reference[axis] );
         }
      }
      if (!Device.Buffer.empty()) {
         for (int axis = 0; axis <= Dim; ++axis) {
            if (Device.Buffer[axis] != nullptr) hipFree( Device.Buffer[axis] );
         }
      }
      if (Device.CoordinatesDevicePtr != nullptr) hipFree( Device.CoordinatesDevicePtr );
      if (Device.Root != nullptr) hipFree( Device.Root );
      hipStreamDestroy( Device.Stream );
   }

   void KdtreeCUDA::prepareCUDA()
   {
      int device_num = 0;
      CHECK_CUDA( hipGetDeviceCount( &device_num ) );
      if( device_num <= 0 ) throw std::runtime_error( "cuda device not found\n" );

      Device.ID = 0;
      Device.Buffer.resize( Dim + 2, nullptr );
      Device.Reference.resize( Dim + 2, nullptr );

      CHECK_CUDA( hipSetDevice( Device.ID ) );
      CHECK_CUDA( hipStreamCreate( &Device.Stream ) );
   }

   __global__
   void cuInitialize(KdtreeNode* root, int size)
   {
      const auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      const auto step = static_cast<int>(blockDim.x * gridDim.x);
      for (int i = index; i < size; i += step) {
         root[i].Index = i;
         root[i].ParentIndex = -1;
         root[i].LeftChildIndex = -1;
         root[i].RightChildIndex = -1;
      }
   }

   void KdtreeCUDA::initialize(const node_type* coordinates, int size)
   {
      assert( Device.Root == nullptr );
      assert( Device.CoordinatesDevicePtr == nullptr );

      CHECK_CUDA(
         hipMalloc(
            reinterpret_cast<void**>(&Device.CoordinatesDevicePtr),
            sizeof( node_type ) * Dim * (size + 1)
         )
      );
      CHECK_CUDA(
         hipMemcpyAsync(
            Device.CoordinatesDevicePtr, coordinates, sizeof( node_type ) * Dim * size,
            hipMemcpyHostToDevice, Device.Stream
         )
      );

      node_type max_value[Dim];
      for (int i = 0; i < Dim; ++i) max_value[i] = std::numeric_limits<node_type>::max();
      CHECK_CUDA(
         hipMemcpyAsync(
            Device.CoordinatesDevicePtr + size * Dim, max_value, sizeof( node_type ) * Dim,
            hipMemcpyHostToDevice, Device.Stream
         )
      );

      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.Root), sizeof( KdtreeNode ) * size ) );

      cuInitialize<<<ThreadBlockNum, ThreadNum, 0, Device.Stream>>>( Device.Root, size );
   }

   __global__
   void cuInitializeReference(int* reference, int size)
   {
      const auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      const auto step = static_cast<int>(blockDim.x * gridDim.x);
      for (int i = index; i < size; i += step) {
         reference[i] = i;
      }
   }

   void KdtreeCUDA::initializeReference(int axis)
   {
      std::vector<int*>& references = Device.Reference;
      for (int i = 0; i <= Dim + 1; ++i) {
         if (references[i] == nullptr) {
            CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&references[i]), sizeof( int ) * TupleNum ) );
         }
      }
      cuInitializeReference<<<ThreadBlockNum, ThreadNum, 0, Device.Stream>>>( references[axis], TupleNum );
      CHECK_KERNEL;
   }

   __global__
   void cuCopyCoordinates(
      node_type* target,
      const int* reference,
      const node_type* coordinates,
      int size,
      int axis,
      int dim
   )
   {
      const auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      const auto step = static_cast<int>(blockDim.x * gridDim.x);
      for (int i = index; i < size; i += step) {
         target[i] = coordinates[reference[i] * dim + axis];
      }
   }

   __device__
   node_type compareSuperKey(const node_type* a, const node_type* b, int axis, int dim)
   {
      node_type difference = a[axis] - b[axis];
      for (int i = 1; difference == 0 && i < dim; ++i) {
         int r = i + axis;
         r = r < dim ? r : r - dim;
         difference = a[r] - b[r];
      }
      return difference;
   }

   __device__
   node_type compareSuperKey(const node_type* a, const node_type* b, node_type delta, int axis, int dim)
   {
      node_type difference = a[axis] - (b[axis] + delta);
      for (int i = 1; difference == 0 && i < dim; ++i) {
         int r = i + axis;
         r = r < dim ? r : r - dim;
         difference = a[r] - (b[r] + delta);
      }
      return difference;
   }

   __device__
   node_type compareSuperKey(
      node_type front_a,
      node_type front_b,
      const node_type* a,
      const node_type* b,
      int axis,
      int dim
   )
   {
      node_type difference = front_a - front_b;
      for (int i = 1; difference == 0 && i < dim; ++i) {
         int r = i + axis;
         r = r < dim ? r : r - dim;
         difference = a[r] - b[r];
      }
      return difference;
   }

   __device__
   int search(
      int r,
      node_type v,
      const int* reference,
      const node_type* buffer,
      const node_type* coordinates,
      int length,
      int step,
      int axis,
      int dim,
      bool inclusive
   )
   {
      if (length == 0) return 0;

      // Find the right place to put v among buffer in ascending order.
      // When it is inclusive, the place will be the rightmost among the same values with v.
      // When it is exclusive, the place will be the leftmost among the same values with v.
      // Local variable i points the index to put v, which means the number of values less than (or equal to) v.
      int i = 0;
      while (step > 0) {
         const int j = min( i + step, length );
         const node_type t = compareSuperKey(
            buffer[j - 1], v, coordinates + reference[j - 1] * dim, coordinates + r * dim, axis, dim
         );
         if (t < 0 || (inclusive && t == 0)) i = j;
         step >>= 1;
      }
      return i;
   }

   __global__
   void cuSortByBlock(
      int* target_reference,
      node_type* target_buffer,
      const int* source_reference,
      const node_type* source_buffer,
      const node_type* coordinates,
      int size,
      int axis,
      int dim
   )
   {
      __shared__ int reference[SharedSize];
      __shared__ node_type buffer[SharedSize];

      const auto t = static_cast<int>(threadIdx.x);
      const int target_block_size = static_cast<int>(blockDim.x) * 2;
      const int index = static_cast<int>(blockIdx.x) * target_block_size + t;
      source_buffer += index;
      source_reference += index;
      target_buffer += index;
      target_reference += index;
      buffer[t] = source_buffer[0];
      reference[t] = source_reference[0];
      buffer[blockDim.x + t] = source_buffer[blockDim.x];
      reference[blockDim.x + t] = source_reference[blockDim.x];

      // Given S = SharedSize, for all threads, [base[i], base[i+step]] is
      // step 1: [0, 1] ... [S-2, S-1]
      // step 2: [0, 2] [1, 3] ... [S-4, S-2], [S-3, S-1]
      // step 4: [0, 4] [1, 5] [2, 6] [3, 7] ... [S-8, S-4] [S-7, S-3] [S-6, S-2] [S-5, S-1]
      //   ...
      // step S/2: [0, S/2] ... [S/2-1, S-1]
      for (int step = 1; step < target_block_size; step <<= 1) {
         const int i = t & (step - 1);
         const int offset = (t - i) * 2;
         int* reference_base = reference + offset;
         node_type* buffer_base = buffer + offset;

         // Merge the sorted array X, base[0] ~ base[step-1], and Y, base[step] ~ base[step*2-1]
         __syncthreads();
         const int reference_x = reference_base[i];
         const node_type buffer_x = buffer_base[i];
         const int x = search(
            reference_x, buffer_x, reference_base + step, buffer_base + step, coordinates, step, step, axis, dim, false
         ) + i;
         const int reference_y = reference_base[i + step];
         const node_type buffer_y = buffer_base[i + step];
         const int y = search(
            reference_y, buffer_y, reference_base, buffer_base, coordinates, step, step, axis, dim, true
         ) + i;

         __syncthreads();
         buffer_base[x] = buffer_x;
         buffer_base[y] = buffer_y;
         reference_base[x] = reference_x;
         reference_base[y] = reference_y;
      }

      __syncthreads();
      target_buffer[0] = buffer[t];
      target_reference[0] = reference[t];
      target_buffer[blockDim.x] = buffer[blockDim.x + t];
      target_reference[blockDim.x] = reference[blockDim.x + t];
   }

   __global__
   void cuSortLastBlock(
      int* target_reference,
      node_type* target_buffer,
      const int* source_reference,
      const node_type* source_buffer,
      const node_type* coordinates,
      int sorted_size,
      int size,
      int axis,
      int dim
   )
   {
      const auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x) * sorted_size * 2;
      if (index >= size) return;

      const int mid = min( sorted_size, size - index );
      const int end = min( sorted_size * 2, size - index );
      source_buffer += index;
      source_reference += index;
      target_buffer += index;
      target_reference += index;
      int left = 0, right = mid;
      for (int i = 0; i < end; ++i) {
         const bool take_from_left = left < mid && (right >= end || compareSuperKey(
            coordinates + source_reference[left] * dim, coordinates + source_reference[right] * dim, axis, dim
         ) < 0);
         if (take_from_left) {
            target_buffer[i] = source_buffer[left];
            target_reference[i] = source_reference[left];
            left++;
         }
         else {
            target_buffer[i] = source_buffer[right];
            target_reference[i] = source_reference[right];
            right++;
         }
      }
   }

   __global__
   void cuGenerateSampleRanks(
      int* left_ranks,
      int* right_ranks,
      int* reference,
      node_type* buffer,
      const node_type* coordinates,
      int sorted_size,
      int size,
      int axis,
      int dim,
      int total_thread_num
   )
   {
      const auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      if (index >= total_thread_num) return;

      const int i = index & (sorted_size / SampleStride - 1);
      const int segment_base = (index - i) * SampleStride * 2;
      buffer += segment_base;
      reference += segment_base;
      left_ranks += (index - i) * 2;
      right_ranks += (index - i) * 2;

      const int left_elements = sorted_size;
      const int right_elements = min( sorted_size, size - (segment_base + sorted_size) );
      const int left_sample_num = getSampleNum( left_elements );
      const int right_sample_num = getSampleNum( right_elements );
      if (i < left_sample_num) {
         left_ranks[i] = i * SampleStride;
         right_ranks[i] = search(
            reference[i * SampleStride], buffer[i * SampleStride],
            reference + sorted_size, buffer + sorted_size, coordinates,
            right_elements, getNextPowerOfTwo( right_elements ), axis, dim, false
         );
      }
      if (i < right_sample_num) {
         right_ranks[sorted_size / SampleStride + i] = i * SampleStride;
         left_ranks[sorted_size / SampleStride + i] = search(
            reference[sorted_size + i * SampleStride], buffer[sorted_size + i * SampleStride],
            reference, buffer, coordinates,
            left_elements, getNextPowerOfTwo( left_elements ), axis, dim, true
         );
      }
   }

   __global__
   void cuMergeRanksAndIndices(int* limits, const int* ranks, int sorted_size, int size, int total_thread_num)
   {
      const auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      if (index >= total_thread_num) return;

      const int i = index & (sorted_size / SampleStride - 1);
      const int segment_base = (index - i) * SampleStride * 2;
      ranks += (index - i) * 2;
      limits += (index - i) * 2;

      const int left_elements = sorted_size;
      const int right_elements = min( sorted_size, size - (segment_base + sorted_size) );
      const int left_sample_num = getSampleNum( left_elements );
      const int right_sample_num = getSampleNum( right_elements );
      if (i < left_sample_num) {
         int x = 0;
         if (right_sample_num > 0) {
            for (int s = getNextPowerOfTwo( right_sample_num ); s > 0; s >>= 1) {
               const int j = min( x + s, right_sample_num );
               if (ranks[left_sample_num + j - 1] < ranks[i]) x = j;
            }
         }
         limits[x + i] = ranks[i];
      }
      if (i < right_sample_num) {
         int x = 0;
         if (left_sample_num > 0) {
            for (int s = getNextPowerOfTwo( left_sample_num ); s > 0; s >>= 1) {
               const int j = min( x + s, left_sample_num );
               if (ranks[j - 1] <= ranks[left_sample_num + i]) x = j;
            }
         }
         limits[x + i] = ranks[left_sample_num + i];
      }
   }

   __device__
   void merge(
      int* reference,
      node_type* buffer,
      const node_type* coordinates,
      int left_length,
      int right_length,
      int axis,
      int dim
   )
   {
      const int* left_reference = reference;
      const int* right_reference = reference + SampleStride;
      const node_type* left_buffer = buffer;
      const node_type* right_buffer = buffer + SampleStride;

      int left_index, right_index, x, y;
      node_type left_value, right_value;
      if (threadIdx.x < left_length) {
         left_value = left_buffer[threadIdx.x];
         left_index = left_reference[threadIdx.x];
         x = static_cast<int>(threadIdx.x) + search(
            left_index, left_value, right_reference, right_buffer,
            coordinates, right_length, SampleStride, axis, dim, false
         );
      }
      if (threadIdx.x < right_length) {
         right_value = right_buffer[threadIdx.x];
         right_index = right_reference[threadIdx.x];
         y = static_cast<int>(threadIdx.x) + search(
            right_index, right_value, left_reference, left_buffer,
            coordinates, left_length, SampleStride, axis, dim, true
         );
      }

      __syncthreads();
      if (threadIdx.x < left_length) {
         buffer[x] = left_value;
         reference[x] = left_index;
      }
      if (threadIdx.x < right_length) {
         buffer[y] = right_value;
         reference[y] = right_index;
      }
   }

   __global__
   void cuMergeReferences(
      int* target_reference,
      node_type* target_buffer,
      const int* source_reference,
      const node_type* source_buffer,
      const node_type* coordinates,
      const int* left_limits,
      const int* right_limits,
      int sorted_size,
      int size,
      int axis,
      int dim
   )
   {
      const int i = static_cast<int>(blockIdx.x) & (sorted_size * 2 / SampleStride - 1);
      const int segment_base = (static_cast<int>(blockIdx.x) - i) * SampleStride;
      target_buffer += segment_base;
      target_reference += segment_base;
      source_buffer += segment_base;
      source_reference += segment_base;

      __shared__ int reference[SampleStride * 2];
      __shared__ node_type buffer[SampleStride * 2];
      __shared__ int left_start_source, right_start_source;
      __shared__ int left_start_target, right_start_target;
      __shared__ int left_length, right_length;

      if (threadIdx.x == 0) {
         const int left_elements = sorted_size;
         const int right_elements = min( sorted_size, size - (segment_base + sorted_size) );
         const int sample_num = getSampleNum( left_elements ) + getSampleNum( right_elements );
         const int left_end_source = i < sample_num - 1 ? left_limits[blockIdx.x + 1] : left_elements;
         const int right_end_source = i < sample_num - 1 ? right_limits[blockIdx.x + 1] : right_elements;
         left_start_source = left_limits[blockIdx.x];
         right_start_source = right_limits[blockIdx.x];
         left_length = left_end_source - left_start_source;
         right_length = right_end_source - right_start_source;
         left_start_target = left_start_source + right_start_source;
         right_start_target = left_start_target + left_length;
      }
      __syncthreads();

      if (threadIdx.x < left_length) {
         buffer[threadIdx.x] = source_buffer[left_start_source + threadIdx.x];
         reference[threadIdx.x] = source_reference[left_start_source + threadIdx.x];
      }
      if (threadIdx.x < right_length) {
         buffer[SampleStride + threadIdx.x] = source_buffer[sorted_size + right_start_source + threadIdx.x];
         reference[SampleStride + threadIdx.x] = source_reference[sorted_size + right_start_source + threadIdx.x];
      }
      __syncthreads();

      merge( reference, buffer, coordinates, left_length, right_length, axis, dim );
      __syncthreads();

      if (threadIdx.x < left_length) {
         target_buffer[left_start_target + threadIdx.x] = buffer[threadIdx.x];
         target_reference[left_start_target + threadIdx.x] = reference[threadIdx.x];
      }
      if (threadIdx.x < right_length) {
         target_buffer[right_start_target + threadIdx.x] = buffer[left_length + threadIdx.x];
         target_reference[right_start_target + threadIdx.x] = reference[left_length + threadIdx.x];
      }
   }

   void KdtreeCUDA::sortByAxis(int axis)
   {
      assert( Device.CoordinatesDevicePtr != nullptr );
      assert( Device.Reference[axis] != nullptr && Device.Reference[Dim] != nullptr );

      if (Device.Buffer[axis] == nullptr) {
         CHECK_CUDA(
            hipMalloc( reinterpret_cast<void**>(&Device.Buffer[axis]), sizeof( node_type ) * TupleNum )
         );
         cuCopyCoordinates<<<ThreadBlockNum, ThreadNum, 0, Device.Stream>>>(
            Device.Buffer[axis], Device.Reference[axis],
            Device.CoordinatesDevicePtr, TupleNum, axis, Dim
         );
         CHECK_KERNEL;
      }
      if (Device.Buffer[Dim] == nullptr) {
         CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.Buffer[Dim]), sizeof( node_type ) * TupleNum ) );
      }

      int stage_num = 0;
      int* in_reference = nullptr;
      int* out_reference = nullptr;
      node_type* in_buffer = nullptr;
      node_type* out_buffer = nullptr;
      for (int step = SharedSize; step < TupleNum; step <<= 1) stage_num++;
      if (stage_num & 1) {
         in_buffer = Device.Sort.Buffer;
         in_reference = Device.Sort.Reference;
         out_buffer = Device.Buffer[Dim];
         out_reference = Device.Reference[Dim];
      }
      else {
         in_buffer = Device.Buffer[Dim];
         in_reference = Device.Reference[Dim];
         out_buffer = Device.Sort.Buffer;
         out_reference = Device.Sort.Reference;
      }

      assert( TupleNum <= SampleStride * Device.Sort.MaxSampleNum );

      int block_num = TupleNum / SharedSize;
      if (block_num > 0) {
         cuSortByBlock<<<block_num, SharedSize / 2, 0, Device.Stream>>>(
            in_reference, in_buffer,
            Device.Reference[axis], Device.Buffer[axis], Device.CoordinatesDevicePtr, TupleNum, axis, Dim
         );
         CHECK_KERNEL;
      }
      const int remained_size = TupleNum % SharedSize;
      if (remained_size > 0) {
         int buffer_index = 0;
         const int start_offset = TupleNum - remained_size;
         const std::array<node_type*, 2> buffers{ Device.Buffer[axis] + start_offset, in_buffer + start_offset };
         const std::array<int*, 2> references{ Device.Reference[axis] + start_offset, in_reference + start_offset };
         for (int sorted_size = 1; sorted_size < remained_size; sorted_size <<= 1) {
            cuSortLastBlock<<<divideUp( remained_size, ThreadNum ), ThreadNum, 0, Device.Stream>>>(
               references[buffer_index ^ 1], buffers[buffer_index ^ 1],
               references[buffer_index], buffers[buffer_index], Device.CoordinatesDevicePtr,
               sorted_size, remained_size, axis, Dim
            );
            CHECK_KERNEL;
            buffer_index ^= 1;
         }
         if (buffer_index == 0) {
            CHECK_CUDA(
               hipMemcpyAsync(
                  buffers[1], buffers[0], sizeof( node_type ) * remained_size, hipMemcpyDeviceToDevice, Device.Stream
               )
            );
            CHECK_CUDA(
               hipMemcpyAsync(
                  references[1], references[0], sizeof( int ) * remained_size, hipMemcpyDeviceToDevice, Device.Stream
               )
            );
         }
      }

      for (int sorted_size = SharedSize; sorted_size < TupleNum; sorted_size <<= 1) {
         constexpr int thread_num = SampleStride * 2;
         const int remained_threads = TupleNum % (sorted_size * 2);
         const int total_thread_num = remained_threads > sorted_size ?
            (TupleNum - remained_threads + sorted_size * 2) / thread_num : (TupleNum - remained_threads) / thread_num;
         block_num = divideUp( total_thread_num, thread_num );
         cuGenerateSampleRanks<<<block_num, thread_num, 0, Device.Stream>>>(
            Device.Sort.LeftRanks, Device.Sort.RightRanks,
            in_reference, in_buffer, Device.CoordinatesDevicePtr,
            sorted_size, TupleNum, axis, Dim, total_thread_num
         );
         CHECK_KERNEL;

         cuMergeRanksAndIndices<<<block_num, thread_num, 0, Device.Stream>>>(
            Device.Sort.LeftLimits, Device.Sort.LeftRanks, sorted_size, TupleNum, total_thread_num
         );
         CHECK_KERNEL;

         cuMergeRanksAndIndices<<<block_num, thread_num, 0, Device.Stream>>>(
            Device.Sort.RightLimits, Device.Sort.RightRanks, sorted_size, TupleNum, total_thread_num
         );
         CHECK_KERNEL;

         const int merge_pairs = remained_threads > sorted_size ?
            getSampleNum( TupleNum ) : (TupleNum - remained_threads) / SampleStride;
         cuMergeReferences<<<merge_pairs, SampleStride, 0, Device.Stream>>>(
            out_reference, out_buffer,
            in_reference, in_buffer, Device.CoordinatesDevicePtr,
            Device.Sort.LeftLimits, Device.Sort.RightLimits,
            sorted_size, TupleNum, axis, Dim
         );
         CHECK_KERNEL;

         if (remained_threads <= sorted_size) {
            CHECK_CUDA(
               hipMemcpyAsync(
                  out_reference + TupleNum - remained_threads, in_reference + TupleNum - remained_threads,
                  sizeof( int ) * remained_threads, hipMemcpyDeviceToDevice, Device.Stream
               )
            );
            CHECK_CUDA(
               hipMemcpyAsync(
                  out_buffer + TupleNum - remained_threads, in_buffer + TupleNum - remained_threads,
                  sizeof( node_type ) * remained_threads, hipMemcpyDeviceToDevice, Device.Stream
               )
            );
         }

         std::swap( in_reference, out_reference );
         std::swap( in_buffer, out_buffer );
      }
   }

   __device__ int num_after_removal;
   __device__ int removal_error;

   __global__
   void cuRemoveDuplicates(
      int* unique_num_in_warp,
      int* target_reference,
      node_type* target_buffer,
      const int* source_reference,
      const node_type* source_buffer,
      const node_type* coordinates,
      int size_per_warp,
      int size,
      int axis,
      int dim
   )
   {
      const auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      const int warp_index = index / warpSize;
      const int warp_lane = index & (warpSize - 1);
      const int offset = warp_index * size_per_warp;
      size_per_warp = min( size_per_warp, size - offset );

      __shared__ int reference[SharedSize];
      __shared__ node_type buffer[SharedSize];

      int* out_reference = target_reference + offset;
      node_type* out_buffer = target_buffer + offset;
      const int* in_reference = source_reference + offset;
      const node_type* in_buffer = source_buffer + offset;
      const int warps_per_block = warpSize / 2;
      const int shared_base = warpSize * 2 * (warp_index % warps_per_block);
      const int precede_mask = (1 << warp_lane) - 1;

      node_type t, v;
      int r, processed_size, unique_mask = 0;
      for (processed_size = 0; processed_size < size_per_warp && unique_mask == 0; processed_size += warpSize) {
         if (warp_lane < size_per_warp) {
            buffer[shared_base + warp_lane] = v = in_buffer[warp_lane];
            reference[shared_base + warp_lane] = r = in_reference[warp_lane];
            if (warp_lane > 0) {
               t = compareSuperKey(
                  v, buffer[shared_base + warp_lane - 1],
                  coordinates + r * dim, coordinates + reference[shared_base + warp_lane - 1] * dim,
                  axis, dim
               );
            }
            else if (warp_index > 0) {
               t = compareSuperKey(
                  v, *(in_buffer - 1),
                  coordinates + r * dim, coordinates + *(in_reference - 1) * dim,
                  axis, dim
               );
            }
            else t = 1;
         }
         else t = 0;

         if (t < 0) removal_error = -1;
         in_buffer += warpSize;
         in_reference += warpSize;

         unique_mask = static_cast<int>(__ballot_sync( 0xffffffff, t > 0 ));
         if (t > 0) {
            const int i = __popc( unique_mask & precede_mask );
            buffer[shared_base + i] = v;
            reference[shared_base + i] = r;
         }
      }

      int write_num = __popc( unique_mask );
      if (write_num == warpSize) {
         out_buffer[warp_lane] = buffer[shared_base + warp_lane];
         out_reference[warp_lane] = reference[shared_base + warp_lane];
      }

      const int shared_address_mask = warpSize * 2 - 1;
      for (; processed_size < size_per_warp; processed_size += warpSize) {
         if (processed_size + warp_lane < size_per_warp) {
            const int i = (write_num + warp_lane) & shared_address_mask;
            const int j = (write_num + warp_lane - 1) & shared_address_mask;
            buffer[shared_base + i] = v = in_buffer[warp_lane];
            reference[shared_base + i] = r = in_reference[warp_lane];
            t = compareSuperKey(
               v, buffer[shared_base + j],
               coordinates + r * dim, coordinates + reference[shared_base + j] * dim,
               axis, dim
            );
         }
         else t = 0;

         if (t < 0) removal_error = -1;
         in_buffer += warpSize;
         in_reference += warpSize;

         unique_mask = static_cast<int>(__ballot_sync( 0xffffffff, t > 0 ));
         if (t > 0) {
            const int i = (write_num + __popc( unique_mask & precede_mask )) & shared_address_mask;
            buffer[shared_base + i] = v;
            reference[shared_base + i] = r;
         }

         const int n = __popc( unique_mask );
         if (((write_num ^ (write_num + n)) & warpSize) != 0) {
            const int i = (write_num & ~(warpSize - 1)) + warp_lane;
            out_buffer[i] = buffer[shared_base + (write_num & warpSize) + warp_lane];
            out_reference[i] = reference[shared_base + (write_num & warpSize) + warp_lane];
         }
         write_num += n;
      }

      if (warp_lane < (write_num & (warpSize - 1))) {
         const int i = (write_num & ~(warpSize - 1)) + warp_lane;
         out_buffer[i] = buffer[shared_base + (write_num & warpSize) + warp_lane];
         out_reference[i] = reference[shared_base + (write_num & warpSize) + warp_lane];
      }

      if (warp_lane == 0 && unique_num_in_warp != nullptr) unique_num_in_warp[warp_index] = write_num;
   }

   __global__
   void cuRemoveGaps(
      int* target_reference,
      node_type* target_buffer,
      const int* source_reference,
      const node_type* source_buffer,
      const int* unique_num_in_warp,
      int size_per_warp,
      int size
   )
   {
      const auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      const int warp_index = index / warpSize;
      const int warp_lane = index & (warpSize - 1);
      const int offset = warp_index * size_per_warp;

      int target_offset = 0, unique_num_in_this_warp = 0;
      if (warp_lane == 0) {
         for (int i = 0; i < warp_index; ++i) target_offset += unique_num_in_warp[i];
         unique_num_in_this_warp = unique_num_in_warp[warp_index];
      }
      target_offset = __shfl_sync( 0xffffffff, target_offset, 0 );
      unique_num_in_this_warp = __shfl_sync( 0xffffffff, unique_num_in_this_warp, 0 );

      source_buffer += offset;
      source_reference += offset;
      target_buffer += target_offset;
      target_reference += target_offset;
      for (int i = warp_lane; i < unique_num_in_this_warp; i += warpSize) {
         target_buffer[i] = source_buffer[i];
         target_reference[i] = source_reference[i];
      }

      if (warp_lane == 0 && offset + size_per_warp >= size) num_after_removal = target_offset + unique_num_in_this_warp;
   }

   int KdtreeCUDA::removeDuplicates(int axis) const
   {
      const int source_index = Dim;
      const int target_index = axis;

      assert( Device.Buffer[source_index] != nullptr && Device.Buffer[target_index] != nullptr );
      assert( Device.Reference[source_index] != nullptr && Device.Reference[target_index] != nullptr );

      int error = 0;
      CHECK_CUDA(
         hipMemcpyToSymbolAsync(HIP_SYMBOL(
            removal_error), &error, sizeof( removal_error ), 0,
            hipMemcpyHostToDevice, Device.Stream
         )
      );

      constexpr int total_thread_num = ThreadBlockNum * ThreadNum;
      constexpr int block_num = std::max( total_thread_num * 2 / SharedSize, 1 );
      constexpr int thread_num_per_block = std::min( total_thread_num, SharedSize / 2 );
      constexpr int segment = total_thread_num / WarpSize;
      const int size_per_warp = divideUp( TupleNum, segment );

      int* unique_num_in_warp = nullptr;
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&unique_num_in_warp), sizeof( int ) * segment ) );
      cuRemoveDuplicates<<<block_num, thread_num_per_block, 0, Device.Stream>>>(
         unique_num_in_warp, Device.Sort.Reference, Device.Sort.Buffer,
         Device.Reference[source_index], Device.Buffer[source_index],
         Device.CoordinatesDevicePtr, size_per_warp, TupleNum, axis, Dim
      );
      CHECK_KERNEL;

      cuRemoveGaps<<<block_num, thread_num_per_block, 0, Device.Stream>>>(
         Device.Reference[target_index], Device.Buffer[target_index],
         Device.Sort.Reference, Device.Sort.Buffer,
         unique_num_in_warp, size_per_warp, TupleNum
      );
      CHECK_KERNEL;

      CHECK_CUDA( hipFree( unique_num_in_warp ) );

      CHECK_CUDA(
         hipMemcpyFromSymbolAsync( &error, HIP_SYMBOL(removal_error), sizeof( error ), 0, hipMemcpyDeviceToHost, Device.Stream )
      );
      if (error != 0) {
         std::ostringstream buffer;
         buffer << "error in removeDuplicates(): " << error << "\n";
         throw std::runtime_error( buffer.str() );
      }

      int num = 0;
      CHECK_CUDA(
         hipMemcpyFromSymbolAsync( &num, HIP_SYMBOL(num_after_removal), sizeof( num ), 0, hipMemcpyDeviceToHost, Device.Stream )
      );
      return num;
   }

   void KdtreeCUDA::sort(std::vector<int>& end)
   {
      const int max_sample_num = TupleNum / SampleStride + 1;
      Device.Sort.MaxSampleNum = max_sample_num;
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.Sort.LeftRanks), sizeof( int ) * max_sample_num ) );
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.Sort.RightRanks), sizeof( int ) * max_sample_num ) );
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.Sort.LeftLimits), sizeof( int ) * max_sample_num ) );
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.Sort.RightLimits), sizeof( int ) * max_sample_num ) );
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.Sort.Reference), sizeof( int ) * TupleNum ) );
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.Sort.Buffer), sizeof( node_type ) * TupleNum ) );

      for (int axis = 0; axis < Dim; ++axis) {
         initializeReference( axis );
         sortByAxis( axis );
         end[axis] = removeDuplicates( axis );
      }
      Device.TupleNum = end[0];
      CHECK_CUDA( hipStreamSynchronize( Device.Stream ) );

      CHECK_CUDA( hipFree( Device.Sort.LeftRanks ) );
      CHECK_CUDA( hipFree( Device.Sort.RightRanks ) );
      CHECK_CUDA( hipFree( Device.Sort.LeftLimits ) );
      CHECK_CUDA( hipFree( Device.Sort.RightLimits ) );
      CHECK_CUDA( hipFree( Device.Sort.Reference ) );
      CHECK_CUDA( hipFree( Device.Sort.Buffer ) );
      for (int axis = 0; axis <= Dim; ++axis) CHECK_CUDA( hipFree( Device.Buffer[axis] ) );
   }

   __device__
   void partition(
      int* target_left_reference,
      int* target_right_reference,
      int* left_child_num_in_warp,
      int* right_child_num_in_warp,
      const int* source_reference,
      const node_type* __restrict__ coordinates,
      int mid_reference,
      int size_per_warp,
      int partition_size,
      int axis,
      int dim,
      int warp_num_per_node
   )
   {
      const auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      const int warp_index = (index / warpSize) & (warp_num_per_node - 1);
      const int warp_lane = index & (warpSize - 1);
      const int offset = warp_index * size_per_warp;
      size_per_warp = min( size_per_warp, partition_size - offset );

      __shared__ int left_reference[SharedSize];
      __shared__ int right_reference[SharedSize];

      int* out_left_reference = target_left_reference + offset;
      int* out_right_reference = target_right_reference + offset;
      const int* in_reference = source_reference + offset;
      const int warps_per_block = warpSize / 2;
      const int shared_base = warpSize * 2 * ((index / warpSize) % warps_per_block);
      const int shared_address_mask = warpSize * 2 - 1;
      const int precede_mask = (1 << warp_lane) - 1;

      node_type t;
      int r, left_write_num = 0, right_write_num = 0;
      for (int processed_size = 0; processed_size < size_per_warp; processed_size += warpSize) {
         if (processed_size + warp_lane < size_per_warp) {
            r = in_reference[warp_lane];
            t = compareSuperKey(
               coordinates[r * dim + axis], coordinates[mid_reference * dim + axis],
               coordinates + r * dim, coordinates + mid_reference * dim, axis, dim
            );
         }
         else t = 0;
         in_reference += warpSize;

         int unique_mask = static_cast<int>(__ballot_sync( 0xffffffff, t < 0 ));
         if (t < 0) {
            const int i = (left_write_num + __popc( unique_mask & precede_mask )) & shared_address_mask;
            left_reference[shared_base + i] = r;
         }

         int n = __popc( unique_mask );
         if (((left_write_num ^ (left_write_num + n)) & warpSize) != 0) {
            const int i = (left_write_num & ~(warpSize - 1)) + warp_lane;
            out_left_reference[i] = left_reference[shared_base + (left_write_num & warpSize) + warp_lane];
         }
         left_write_num += n;

         unique_mask = static_cast<int>(__ballot_sync( 0xffffffff, t > 0 ));
         if (t > 0) {
            const int i = (right_write_num + __popc( unique_mask & precede_mask )) & shared_address_mask;
            right_reference[shared_base + i] = r;
         }

         n = __popc( unique_mask );
         if (((right_write_num ^ (right_write_num + n)) & warpSize) != 0) {
            const int i = (right_write_num & ~(warpSize - 1)) + warp_lane;
            out_right_reference[i] = right_reference[shared_base + (right_write_num & warpSize) + warp_lane];
         }
         right_write_num += n;
      }

      if (warp_lane < (left_write_num & (warpSize - 1))) {
         const int i = (left_write_num & ~(warpSize - 1)) + warp_lane;
         out_left_reference[i] = left_reference[shared_base + (left_write_num & warpSize) + warp_lane];
      }
      if (warp_lane < (right_write_num & (warpSize - 1))) {
         const int i = (right_write_num & ~(warpSize - 1)) + warp_lane;
         out_right_reference[i] = right_reference[shared_base + (right_write_num & warpSize) + warp_lane];
      }

      if (warp_lane == 0 && left_child_num_in_warp != nullptr) left_child_num_in_warp[warp_index] = left_write_num;
      if (warp_lane == 0 && right_child_num_in_warp != nullptr) right_child_num_in_warp[warp_index] = right_write_num;
   }

   __global__
   void cuPartition(
      KdtreeNode* root,
      int* left_child_num_in_warp,
      int* right_child_num_in_warp,
      int* target_left_reference,
      int* target_right_reference,
      int* mid_references,
      const int* last_mid_references,
      const int* source_reference,
      const int* primary_reference,
      const node_type* __restrict__ coordinates,
      int start,
      int end,
      int axis,
      int dim,
      int depth
   )
   {
      const auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      const auto total_warp_num = static_cast<int>(blockDim.x * gridDim.x / warpSize);
      const int warp_num_per_node = total_warp_num >> depth;
      const int warp_index = index / warpSize;
      const int warp_lane = index & (warpSize - 1);

      int mid = start + (end - start) / 2;
      for (int i = 1; i <= depth; ++i) {
         if (warp_index & (total_warp_num >> i)) start = mid + 1;
         else end = mid - 1;
         mid = start + (end - start) / 2;
      }

      const int partition_size = end - start + 1;
      const int size_per_warp = divideUp( partition_size, warp_num_per_node );
      const int mid_reference = primary_reference[mid];
      partition(
         target_left_reference + start, target_right_reference + start,
         left_child_num_in_warp + (warp_index & ~(warp_num_per_node - 1)),
         right_child_num_in_warp + (warp_index & ~(warp_num_per_node - 1)),
         source_reference + start, coordinates,
         mid_reference, size_per_warp, partition_size, axis, dim, warp_num_per_node
      );

      if (warp_lane == 0) {
         const int m = warp_index / warp_num_per_node;
         mid_references[m] = mid_reference;
         if (last_mid_references != nullptr) {
            if (m & 1) root[last_mid_references[m >> 1]].RightChildIndex = mid_reference;
            else root[last_mid_references[m >> 1]].LeftChildIndex = mid_reference;
            root[mid_reference].ParentIndex = root[last_mid_references[m >> 1]].Index;
         }
      }
   }

   __global__
   void cuRemovePartitionGaps(
      int* target_reference,
      const int* source_left_reference,
      const int* source_right_reference,
      const int* left_child_num_in_warp,
      const int* right_child_num_in_warp,
      int start,
      int end,
      int depth
   )
   {
      const auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      const auto total_warp_num = static_cast<int>(blockDim.x * gridDim.x / warpSize);
      const int warp_num_per_node = total_warp_num >> depth;
      const int warp_lane = index & (warpSize - 1);
      const int warp_index = index / warpSize;

      int mid = start + (end - start) / 2;
      for (int i = 1; i <= depth; ++i) {
         if (warp_index & (total_warp_num >> i)) start = mid + 1;
         else end = mid - 1;
         mid = start + (end - start) / 2;
      }

      const int partition_size = end - start + 1;
      const int size_per_warp = divideUp( partition_size, warp_num_per_node );
      const int offset = start + size_per_warp * (warp_index - (warp_index & ~(warp_num_per_node - 1)));

      int target_offset = start, child_num_in_this_warp = 0;
      if (warp_lane == 0) {
         for (int i = warp_index & ~(warp_num_per_node - 1); i < warp_index; ++i) {
            target_offset += left_child_num_in_warp[i];
         }
         child_num_in_this_warp = left_child_num_in_warp[warp_index];
      }
      target_offset = __shfl_sync( 0xffffffff, target_offset, 0 );
      child_num_in_this_warp = __shfl_sync( 0xffffffff, child_num_in_this_warp, 0 );
      for (int i = warp_lane; i < child_num_in_this_warp; i += warpSize) {
         target_reference[target_offset + i] = source_left_reference[offset + i];
      }

      target_offset = mid + 1;
      if (warp_lane == 0) {
         for (int i = warp_index & ~(warp_num_per_node - 1); i < warp_index; ++i) {
            target_offset += right_child_num_in_warp[i];
         }
         child_num_in_this_warp = right_child_num_in_warp[warp_index];
      }
      target_offset = __shfl_sync( 0xffffffff, target_offset, 0 );
      child_num_in_this_warp = __shfl_sync( 0xffffffff, child_num_in_this_warp, 0 );
      for (int i = warp_lane; i < child_num_in_this_warp; i += warpSize) {
         target_reference[target_offset + i] = source_right_reference[offset + i];
      }
   }

   __global__
   void cuPartition(
      KdtreeNode* root,
      int* target_reference,
      int* mid_references,
      const int* last_mid_references,
      const int* source_reference,
      const int* primary_reference,
      const node_type* __restrict__ coordinates,
      int start,
      int end,
      int axis,
      int dim,
      int depth,
      int max_controllable_depth_for_warp
   )
   {
      const auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      const auto total_warp_num = static_cast<int>(blockDim.x * gridDim.x / warpSize);
      const int warp_lane = index & (warpSize - 1);
      const int warp_index = index / warpSize;
      const int loop_levels = depth - max_controllable_depth_for_warp;
      for (int loop = 0; loop < (1 << loop_levels); ++loop) {
         int s = start, e = end, mid;
         for (int i = 1; i <= loop_levels; ++i) {
            mid = s + (e - s) / 2;
            if (loop & (1 << (loop_levels - i))) s = mid + 1;
            else e = mid - 1;
         }
         for (int i = 1; i <= max_controllable_depth_for_warp; ++i) {
            mid = s + (e - s) / 2;
            if (warp_index & (total_warp_num >> i)) s = mid + 1;
            else e = mid - 1;
         }
         mid = s + (e - s) / 2;

         const int partition_size = e - s + 1;
         const int mid_reference = primary_reference[mid];
         partition(
            target_reference + s, target_reference + mid + 1, nullptr, nullptr,
            source_reference + s, coordinates,
            mid_reference, partition_size, partition_size, axis, dim, 1
         );

         if (warp_lane == 0) {
            const int m = warp_index + total_warp_num * loop;
            mid_references[m] = mid_reference;
            if (last_mid_references != nullptr) {
               if (m & 1) root[last_mid_references[m >> 1]].RightChildIndex = mid_reference;
               else root[last_mid_references[m >> 1]].LeftChildIndex = mid_reference;
               root[mid_reference].ParentIndex = root[last_mid_references[m >> 1]].Index;
            }
         }
      }
   }

   __global__
   void cuCopyReference(int* target_reference, const int* source_reference, int size)
   {
      const auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      const auto step = static_cast<int>(blockDim.x * gridDim.x);
      for (int i = index; i < size; i += step) target_reference[i] = source_reference[i];
   }

   void KdtreeCUDA::partitionDimension(int axis, int depth)
   {
      constexpr int total_thread_num = ThreadBlockNum * ThreadNum;
      constexpr int block_num = std::max( total_thread_num * 2 / SharedSize, 1 );
      constexpr int thread_num_per_block = std::min( total_thread_num, SharedSize / 2 );
      constexpr int warp_num = total_thread_num / WarpSize;
      const auto max_controllable_depth_for_warp =
         static_cast<int>(std::floor( std::log2( static_cast<double>(warp_num) ) ));

      int* mid_references = Device.MidReferences[depth & 1];
      const int* last_mid_references = depth == 0 ? nullptr : Device.MidReferences[(depth - 1) & 1];
      if (depth < max_controllable_depth_for_warp) {
         for (int i = 1; i < Dim; ++i) {
            int r = i + axis;
            r = r < Dim ? r : r - Dim;
            cuPartition<<<block_num, thread_num_per_block, 0, Device.Stream>>>(
               Device.Root, Device.LeftChildNumInWarp, Device.RightChildNumInWarp,
               Device.Reference[Dim], Device.Reference[Dim + 1], mid_references,
               last_mid_references, Device.Reference[r], Device.Reference[axis],
               Device.CoordinatesDevicePtr, 0, Device.TupleNum - 1, axis, Dim, depth
            );
            CHECK_KERNEL;

            cuRemovePartitionGaps<<<block_num, thread_num_per_block, 0, Device.Stream>>>(
               Device.Reference[r],
               Device.Reference[Dim], Device.Reference[Dim + 1],
               Device.LeftChildNumInWarp, Device.RightChildNumInWarp,
               0, Device.TupleNum - 1, depth
            );
            CHECK_KERNEL;
         }
      }
      else {
         for (int i = 1; i < Dim; ++i) {
            int r = i + axis;
            r = r < Dim ? r : r - Dim;
            cuPartition<<<block_num, thread_num_per_block, 0, Device.Stream>>>(
               Device.Root, Device.Reference[Dim], mid_references,
               last_mid_references, Device.Reference[r], Device.Reference[axis],
               Device.CoordinatesDevicePtr, 0, Device.TupleNum - 1, axis, Dim, depth, max_controllable_depth_for_warp
            );
            CHECK_KERNEL;

            cuCopyReference<<<block_num, thread_num_per_block, 0, Device.Stream>>>(
               Device.Reference[r], Device.Reference[Dim], Device.TupleNum
            );
            CHECK_KERNEL;
         }
      }

      if (depth == 0) {
         CHECK_CUDA(
            hipMemcpyAsync(
               &Device.RootNode, Device.MidReferences[0], sizeof( int ), hipMemcpyDeviceToHost, Device.Stream
            )
         );
         assert( Device.RootNode != -1 );
      }
   }

   __global__
   void cuPartitionFinal(
      KdtreeNode* root,
      int* mid_references,
      const int* last_mid_references,
      const int* primary_reference,
      int start,
      int end,
      int depth
   )
   {
      const auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      const auto total_warp_num = static_cast<int>(blockDim.x * gridDim.x / warpSize);
      const int warp_index = index / warpSize;

      for (int i = 1; i <= depth; ++i) {
         const int mid = start + (end - start) / 2;
         if (warp_index & (total_warp_num >> i)) start = mid + 1;
         else end = mid - 1;
      }

      int mid_reference = -1;
      if (end == start) mid_reference = primary_reference[end];
      else if (end == start + 1) {
         mid_reference = primary_reference[start];
         root[mid_reference].RightChildIndex = primary_reference[end];
         root[root[mid_reference].RightChildIndex].ParentIndex = root[mid_reference].Index;
      }
      else if (end == start + 2) {
         mid_reference = primary_reference[start + 1];
         root[mid_reference].LeftChildIndex = primary_reference[start];
         root[mid_reference].RightChildIndex = primary_reference[end];
         root[root[mid_reference].LeftChildIndex].ParentIndex = root[mid_reference].Index;
         root[root[mid_reference].RightChildIndex].ParentIndex = root[mid_reference].Index;
      }

      if (mid_reference != -1) {
         const int warp_num_per_node = total_warp_num >> depth;
         const int m = warp_index / warp_num_per_node;
         mid_references[m] = mid_reference;
         if (m & 1) root[last_mid_references[m >> 1]].RightChildIndex = mid_reference;
         else root[last_mid_references[m >> 1]].LeftChildIndex = mid_reference;
         root[mid_reference].ParentIndex = root[last_mid_references[m >> 1]].Index;
      }
   }

   void KdtreeCUDA::partitionDimensionFinal(int axis, int depth)
   {
      constexpr int total_thread_num = ThreadBlockNum * ThreadNum;
      constexpr int block_num = std::max( total_thread_num * 2 / SharedSize, 1 );
      constexpr int thread_num_per_block = std::min( total_thread_num, SharedSize / 2 );
      constexpr int warp_num = total_thread_num / WarpSize;
      const auto max_controllable_depth_for_warp =
         static_cast<int>(std::floor( std::log2( static_cast<double>(warp_num) ) ));
      const int loop_levels = std::max( depth - max_controllable_depth_for_warp, 0 );

      int* mid_references = Device.MidReferences[depth & 1];
      const int* last_mid_references = Device.MidReferences[(depth - 1) & 1];
      for (int loop = 0; loop < (1 << loop_levels); ++loop) {
         int start = 0, end = Device.TupleNum - 1;
         for (int i = 1; i <= loop_levels; ++i) {
            const int mid = start + (end - start) / 2;
            if (loop & (1 << (loop_levels - i))) start = mid + 1;
            else end = mid - 1;
         }

         cuPartitionFinal<<<block_num, thread_num_per_block, 0, Device.Stream>>>(
            Device.Root,
            mid_references + loop * warp_num,
            last_mid_references + loop * warp_num / 2,
            Device.Reference[axis],
            start, end, depth - loop_levels
         );
         CHECK_KERNEL;
      }
   }

   void KdtreeCUDA::build()
   {
      constexpr int warp_num = ThreadBlockNum * ThreadNum / WarpSize;
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.LeftChildNumInWarp), sizeof( int ) * warp_num ) );
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.RightChildNumInWarp), sizeof( int ) * warp_num ) );
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.MidReferences[0]), sizeof( int ) * Device.TupleNum ) );
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.MidReferences[1]), sizeof( int ) * Device.TupleNum ) );

      assert( !Device.Reference.empty() );
      for (int axis = 0; axis < Dim; ++axis) assert( Device.Reference[axis] != nullptr );

      if (Device.Reference[Dim] == nullptr) {
         CHECK_CUDA(
            hipMalloc( reinterpret_cast<void**>(&Device.Reference[Dim]), sizeof( int ) * Device.TupleNum )
         );
      }

      const auto depth = static_cast<int>(std::floor( std::log2( static_cast<double>(Device.TupleNum) ) ));
      for (int i = 0; i < depth - 1; ++i) {
         partitionDimension( i % Dim, i );
      }
      partitionDimensionFinal( (depth - 1) % Dim, depth - 1 );

      CHECK_CUDA( hipStreamSynchronize( Device.Stream ) );
      CHECK_CUDA( hipFree( Device.LeftChildNumInWarp ) );
      CHECK_CUDA( hipFree( Device.RightChildNumInWarp ) );
      CHECK_CUDA( hipFree( Device.MidReferences[0] ) );
      CHECK_CUDA( hipFree( Device.MidReferences[1] ) );
   }

   __device__ int verify_error;

   __global__
   void cuVerify(
      int* node_sums,
      int* next_child,
      const int* child,
      const KdtreeNode* root,
      const node_type* coordinates,
      int size,
      int axis,
      int dim
   )
   {
      __shared__ int sums[SharedSize];

      const auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      const auto step = static_cast<int>(blockDim.x * gridDim.x);
      const auto id = static_cast<int>(threadIdx.x);

      int node, count = 0;
      for (int i = index; i < size; i += step) {
         node = child[i];
         if (node >= 0) {
            count++;
            const int right = root[node].RightChildIndex;
            next_child[i * 2 + 1] = right;
            if (right >= 0) {
               if (compareSuperKey(
                     coordinates + root[right].Index * dim, coordinates + root[node].Index * dim, axis, dim
                   ) <= 0) {
                  verify_error = 1;
               }
               if (root[right].ParentIndex != root[node].Index) verify_error = 1;
            }

            const int left = root[node].LeftChildIndex;
            next_child[i * 2] = left;
            if (left >= 0) {
               if (compareSuperKey(
                     coordinates + root[left].Index * dim, coordinates + root[node].Index * dim, axis, dim
                   ) >= 0) {
                  verify_error = 1;
               }
               if (root[left].ParentIndex != root[node].Index) verify_error = 1;
            }
         }
         else next_child[i * 2] = next_child[i * 2 + 1] = -1;
      }
      sums[id] = count;
      __syncthreads();

      for (int i = static_cast<int>(blockDim.x / 2); i > warpSize; i >>= 1) {
         if (id < i) {
            count += sums[id + i];
            sums[id] = count;
         }
         __syncthreads();
      }

      if (id < warpSize) {
         if (blockDim.x >= warpSize * 2) count += sums[id + warpSize];
         for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
            count += __shfl_down_sync( 0xffffffff, count, offset );
         }
      }

      if (id == 0) node_sums[blockIdx.x] += count;
   }

   __global__
   void cuSumNodeNum(int* node_sums)
   {
      __shared__ int sums[SharedSize];

      const auto step = static_cast<int>(blockDim.x * gridDim.x);
      const auto id = static_cast<int>(threadIdx.x);

      int sum = 0;
      for (int i = id; i < ThreadBlockNum; i += step) sum += node_sums[i];
      sums[id] = sum;
      __syncthreads();

      for (int i = static_cast<int>(blockDim.x / 2); i > warpSize; i >>= 1) {
         if (id < i) {
            sum += sums[id + i];
            sums[id] = sum;
         }
         __syncthreads();
      }

      if (id < warpSize) {
         if (blockDim.x >= warpSize * 2) sum += sums[id + warpSize];
         for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
            sum += __shfl_down_sync( 0xffffffff, sum, offset );
         }
      }

      if (id == 0) node_sums[blockIdx.x] = sum;
   }

   int KdtreeCUDA::verify(int start_axis) const
   {
      const auto log_size = static_cast<int>(std::floor( std::log2( static_cast<double>(Device.TupleNum) ) ));
      CHECK_CUDA(
         hipMemcpyAsync(
            Device.MidReferences[0], &Device.RootNode, sizeof( int ), hipMemcpyHostToDevice, Device.Stream
         )
      );

      int error = 0;
      CHECK_CUDA(
         hipMemcpyToSymbolAsync(HIP_SYMBOL( verify_error), &error, sizeof( error ), 0, hipMemcpyHostToDevice, Device.Stream )
      );

      int* child;
      int* next_child;
      for (int i = 0; i <= log_size; ++i) {
         const int needed_threads = 1 << i;
         const int block_num = std::clamp( needed_threads / ThreadNum, 1, ThreadBlockNum );
         const int axis = (i + start_axis) % Dim;
         child = Device.MidReferences[i & 1];
         next_child = Device.MidReferences[(i + 1) & 1];
         cuVerify<<<block_num, ThreadNum, 0, Device.Stream>>>(
            Device.NodeSums, next_child,
            child, Device.Root, Device.CoordinatesDevicePtr, needed_threads, axis, Dim
         );
         CHECK_KERNEL;

         CHECK_CUDA(
            hipMemcpyFromSymbolAsync( &error, HIP_SYMBOL(verify_error), sizeof( error ), 0, hipMemcpyDeviceToHost, Device.Stream )
         );
         CHECK_CUDA( hipStreamSynchronize( Device.Stream ) );
         if (error != 0) {
            std::ostringstream buffer;
            buffer << "error in verify(): " << error << "\n";
            throw std::runtime_error( buffer.str() );
         }
      }

      cuSumNodeNum<<<1, ThreadNum, 0, Device.Stream>>>( Device.NodeSums );
      CHECK_KERNEL;

      int node_num = 0;
      CHECK_CUDA( hipMemcpyAsync( &node_num, Device.NodeSums, sizeof( int ), hipMemcpyDeviceToHost, Device.Stream ) );
      std::cout << node_num << std::endl;
      return node_num;
   }

   int KdtreeCUDA::verify()
   {
      CHECK_CUDA(
         hipMalloc( reinterpret_cast<void**>(&Device.MidReferences[0]), sizeof( int ) * 2 * Device.TupleNum )
      );
      CHECK_CUDA(
         hipMalloc( reinterpret_cast<void**>(&Device.MidReferences[1]), sizeof( int ) * 2 * Device.TupleNum )
      );
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.NodeSums), sizeof( int ) * ThreadBlockNum ) );
      CHECK_CUDA( hipMemset( Device.NodeSums, 0, sizeof( int ) * ThreadBlockNum ) );

      const int node_num = verify( 0 );
      CHECK_CUDA( hipStreamSynchronize( Device.Stream ) );
      CHECK_CUDA( hipFree( Device.MidReferences[0] ) );
      CHECK_CUDA( hipFree( Device.MidReferences[1] ) );
      CHECK_CUDA( hipFree( Device.NodeSums ) );
      return node_num;
   }

   void KdtreeCUDA::create()
   {
      initialize( Coordinates, TupleNum );
      CHECK_CUDA( hipStreamSynchronize( Device.Stream ) );

      auto start_time = std::chrono::steady_clock::now();
      std::vector<int> end(Dim);
      sort( end );
      auto end_time = std::chrono::steady_clock::now();
      const auto sort_time =
         static_cast<double>(std::chrono::duration_cast<std::chrono::nanoseconds>(end_time - start_time).count()) * 1e-9;

      for (int i = 0; i < Dim - 1; ++i) {
         assert( end[i] >= 0 );
         for (int j = i + 1; j < Dim; ++j) assert( end[i] == end[j] );
      }

      start_time = std::chrono::steady_clock::now();
      build();
      end_time = std::chrono::steady_clock::now();
      const auto build_time =
         static_cast<double>(std::chrono::duration_cast<std::chrono::nanoseconds>(end_time - start_time).count()) * 1e-9;

      start_time = std::chrono::steady_clock::now();
      NodeNum = verify();
      end_time = std::chrono::steady_clock::now();
      const auto verify_time =
         static_cast<double>(std::chrono::duration_cast<std::chrono::nanoseconds>(end_time - start_time).count()) * 1e-9;

      std::cout << " >> " << TupleNum - end[0] << " duplicates removed\n";
      std::cout << " >> Number of nodes = " << NodeNum << "\n" << std::fixed << std::setprecision( 2 )
         << " >> Total Time = "  << sort_time + build_time + verify_time << " sec."
         << "\n\t* Sort Time = " << sort_time << " sec."
         << "\n\t* Build Time = " << build_time << " sec."
         << "\n\t* Verify Time = " << verify_time << " sec.\n\n";
   }

   void KdtreeCUDA::print(const std::vector<KdtreeNode>& kd_nodes, int index, int depth) const
   {
      if (kd_nodes[index].RightChildIndex >= 0) print( kd_nodes, kd_nodes[index].RightChildIndex, depth + 1 );

      for (int i = 0; i < depth; ++i) std::cout << "       ";

      const node_type* tuple = Coordinates + kd_nodes[index].Index * Dim;
      std::cout << "(" << tuple[0] << ",";
      for (int i = 1; i < Dim - 1; ++i) std::cout << tuple[i] << ",";
      std::cout << tuple[Dim - 1] << ")\n";

      if (kd_nodes[index].LeftChildIndex >= 0) print( kd_nodes, kd_nodes[index].LeftChildIndex, depth + 1 );
   }

   void KdtreeCUDA::print() const
   {
      if (Device.RootNode < 0 || Coordinates == nullptr) return;

      std::vector<KdtreeNode> kd_nodes(TupleNum);
      CHECK_CUDA(
         hipMemcpyAsync(
            kd_nodes.data(), Device.Root, sizeof( KdtreeNode ) * TupleNum, hipMemcpyDeviceToHost, Device.Stream
         )
      );

      print( kd_nodes, Device.RootNode, 0 );
   }

   void KdtreeCUDA::getResult(
      std::vector<node_type>& output,
      const std::vector<KdtreeNode>& kd_nodes,
      int index,
      int depth
   ) const
   {
      if (kd_nodes[index].RightChildIndex >= 0) {
         getResult( output, kd_nodes, kd_nodes[index].RightChildIndex, depth + 1 );
      }

      const node_type* tuple = Coordinates + kd_nodes[index].Index * Dim;
      output.emplace_back( tuple[0] );
      for (int i = 1; i < Dim - 1; ++i) output.emplace_back( tuple[i] );
      output.emplace_back( tuple[Dim - 1] );

      if (kd_nodes[index].LeftChildIndex >= 0) getResult( output, kd_nodes, kd_nodes[index].LeftChildIndex, depth + 1 );
   }

   void KdtreeCUDA::getResult(std::vector<node_type>& output) const
   {
      if (Device.RootNode < 0 || Coordinates == nullptr) return;

      std::vector<KdtreeNode> kd_nodes(TupleNum);
      CHECK_CUDA(
         hipMemcpyAsync(
            kd_nodes.data(), Device.Root, sizeof( KdtreeNode ) * TupleNum, hipMemcpyDeviceToHost, Device.Stream
         )
      );

      getResult( output, kd_nodes, Device.RootNode, 0 );
   }

   __device__
   void findQueryWithoutStack(
      int* lists,
      int* list_lengths,
      const KdtreeNode* root,
      const node_type* coordinates,
      const node_type* queries,
      node_type search_radius,
      int node_index,
      int query_num,
      int size,
      int dim
   )
   {
      const auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      if (index >= query_num) return;

      int depth = 0;
      int prev = -1;
      int curr = node_index;
      while (curr >= 0) {
         const KdtreeNode* node = &root[curr];
         const int parent = node->ParentIndex;
         if (curr >= size) {
            prev = curr;
            curr = parent;
            continue;
         }

         const node_type* query = queries + index * dim;
         const node_type* point = coordinates + root[curr].Index * dim;
         const bool from_child = prev >= 0 && (prev == node->LeftChildIndex || prev == node->RightChildIndex);
         if (!from_child) {
            bool inside = true;
            for (int d = 0; d < dim; ++d) {
               if (query[d] - search_radius > point[d] || query[d] + search_radius < point[d]) {
                  inside = false;
                  break;
               }
            }
            if (inside) {
               lists[index * size + list_lengths[index]] = node->Index;
               list_lengths[index]++;
            }
         }

         const int axis = depth % dim;
         const bool search_left = node->LeftChildIndex >= 0 &&
            compareSuperKey( point, query, -search_radius, axis, dim ) >= 0;
         const bool search_right = node->RightChildIndex >= 0 &&
            compareSuperKey( point, query, search_radius, axis, dim ) <= 0;

         int next = -1;
         if (search_left && prev == node->LeftChildIndex) {
            if (search_right) {
               next = node->RightChildIndex;
               depth++;
            }
            else {
               next = parent;
               depth--;
            }
         }
         else if (search_right && prev == node->RightChildIndex) {
            next = parent;
            depth--;
         }
         else if (prev < 0 || prev == parent) {
            if (!search_left && !search_right) {
               next = parent;
               depth--;
            }
            else if (!search_left) {
               next = node->RightChildIndex;
               depth++;
            }
            else {
               next = node->LeftChildIndex;
               depth++;
            }
         }

         prev = curr;
         curr = next;
      }
   }

   __device__
   void findQuery(
      int* lists,
      int* list_lengths,
      const KdtreeNode* root,
      const node_type* coordinates,
      const node_type* queries,
      node_type search_radius,
      int node_index,
      int size,
      int dim,
      int mask
   )
   {
      const auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      const int warp_lane = index & (warpSize - 1);
      const int max_node_num_in_stack = SharedSize * 8 + 1;

      int list_ptr = 0, depth = 0;
      int mask_list[2][max_node_num_in_stack];
      const KdtreeNode* visit_list[2][max_node_num_in_stack];
      mask_list[list_ptr][0] = mask;
      visit_list[list_ptr][0] = &root[node_index];
      visit_list[list_ptr][1] = nullptr;
      while (visit_list[list_ptr][0] != nullptr) {
         int child_num = 0;
         const int axis = depth % dim;
         for (int i = 0; visit_list[list_ptr][i] != nullptr; ++i) {
            bool search_left = false, search_right = false;
            const bool active = (mask_list[list_ptr][i] >> warp_lane) & 1;
            if (active) {
               bool inside = true;
               const node_type* query = queries + index * dim;
               const node_type* node = coordinates + visit_list[list_ptr][i]->Index * dim;
               for (int d = 0; d < dim; ++d) {
                  if (query[d] - search_radius > node[d] || query[d] + search_radius < node[d]) {
                     inside = false;
                     break;
                  }
               }
               if (inside) {
                  lists[index * size + list_lengths[index]] = visit_list[list_ptr][i]->Index;
                  list_lengths[index]++;
               }

               search_left = visit_list[list_ptr][i]->LeftChildIndex >= 0 &&
                  compareSuperKey( node, query, -search_radius, axis, dim ) >= 0;
               search_right = visit_list[list_ptr][i]->RightChildIndex >= 0 &&
                  compareSuperKey( node, query, search_radius, axis, dim ) <= 0;
            }

            const int left_mask = static_cast<int>(__ballot_sync( 0xffffffff, search_left ));
            if (__popc( left_mask ) != 0) {
               mask_list[list_ptr ^ 1][child_num] = left_mask;
               visit_list[list_ptr ^ 1][child_num++] = visit_list[list_ptr][i]->LeftChildIndex >= 0 ?
                  &root[visit_list[list_ptr][i]->LeftChildIndex] : &root[node_index];
            }
            const int right_mask = static_cast<int>(__ballot_sync( 0xffffffff, search_right ));
            if (__popc( right_mask ) != 0) {
               mask_list[list_ptr ^ 1][child_num] = right_mask;
               visit_list[list_ptr ^ 1][child_num++] = visit_list[list_ptr][i]->RightChildIndex >= 0 ?
                  &root[visit_list[list_ptr][i]->RightChildIndex] : &root[node_index];
            }
         }
         depth++;
         list_ptr ^= 1;
         visit_list[list_ptr][child_num] = nullptr;
      }
   }

   template<bool small_tree = true>
   __global__
   void cuSearch(
      int* lists,
      int* list_lengths,
      const KdtreeNode* root,
      const node_type* coordinates,
      const node_type* queries,
      node_type search_radius,
      int node_index,
      int query_num,
      int size,
      int dim
   )
   {
      auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      const auto step = static_cast<int>(blockDim.x * gridDim.x);
      while (true) {
         const int mask = static_cast<int>(__ballot_sync( 0xffffffff, index < query_num ));
         if (__popc( mask ) == 0) break;

         if (small_tree) {
            findQuery( lists, list_lengths, root, coordinates, queries, search_radius, node_index, size, dim, mask );
         }
         else {
            findQueryWithoutStack(
               lists, list_lengths,
               root, coordinates, queries, search_radius, node_index, query_num, size, dim
            );
         }
         index += step;
      }
   }

   void KdtreeCUDA::search(
      std::vector<std::vector<int>>& founds,
      const node_type* queries,
      int query_num,
      node_type search_radius
   ) const
   {
      if (Device.RootNode < 0 || Coordinates == nullptr) return;

      int* lists = nullptr;
      int* list_lengths = nullptr;
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&lists), sizeof( int ) * Device.TupleNum * query_num ) );
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&list_lengths), sizeof( int ) * query_num ) );
      CHECK_CUDA( hipMemset( list_lengths, 0, sizeof( int ) * query_num ) );

      node_type* device_queries = nullptr;
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&device_queries), sizeof( node_type ) * query_num * Dim ) );
      CHECK_CUDA(
         hipMemcpyAsync(
            device_queries, queries, sizeof( node_type ) * query_num * Dim,
            hipMemcpyHostToDevice, Device.Stream
         )
      );

      const int max_depth = static_cast<int>(std::log2( static_cast<double>(Device.TupleNum) ));
      const int max_node_num = std::max( 1 << (max_depth - 1), Device.TupleNum - (1 << max_depth) + 1 );
      constexpr int max_node_num_in_stack = SharedSize * 8;
      if (max_node_num <= max_node_num_in_stack) {
         cuSearch<true><<<divideUp( query_num, WarpSize ), WarpSize, 0, Device.Stream>>>(
            lists, list_lengths,
            Device.Root, Device.CoordinatesDevicePtr, device_queries,
            search_radius, Device.RootNode, query_num, Device.TupleNum, Dim
         );
         CHECK_KERNEL;
      }
      else {
         cuSearch<false><<<divideUp( query_num, WarpSize ), WarpSize, 0, Device.Stream>>>(
            lists, list_lengths,
            Device.Root, Device.CoordinatesDevicePtr, device_queries,
            search_radius, Device.RootNode, query_num, Device.TupleNum, Dim
         );
         CHECK_KERNEL;
      }

      std::vector<int> lengths(query_num);
      CHECK_CUDA(
         hipMemcpyAsync(
            lengths.data(), list_lengths, sizeof( int ) * query_num, hipMemcpyDeviceToHost, Device.Stream
         )
      );

      founds.clear();
      founds.resize( query_num );
      for (int i = 0; i < query_num; ++i) {
         if (lengths[i] <= 0) continue;

         founds[i].resize( lengths[i] );
         CHECK_CUDA(
            hipMemcpyAsync(
               founds[i].data(), lists + Device.TupleNum * i, sizeof( int ) * lengths[i],
               hipMemcpyDeviceToHost, Device.Stream
            )
         );
      }

      CHECK_CUDA( hipFree( lists ) );
      CHECK_CUDA( hipFree( list_lengths ) );
      CHECK_CUDA( hipFree( device_queries ) );
   }

   __global__
   void cuInitializeKNN(uint64_t* lists, int query_num, int neighbor_num)
   {
      const auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      const auto step = static_cast<int>(blockDim.x * gridDim.x);
      const uint64_t value =
            (static_cast<uint64_t>(__float_as_uint( INFINITY )) << 32) | static_cast<uint32_t>(-1);
      for (int i = index; i < query_num; i += step) {
         for (int j = 0; j < neighbor_num; ++j) lists[i * neighbor_num + j] = value;
      }
   }

   __device__
   float push(uint64_t* lists, int node_index, int neighbor_num, float squared_distance)
   {
      uint64_t value =
            (static_cast<uint64_t>(__float_as_uint( squared_distance )) << 32) | static_cast<uint32_t>(node_index);
      for (int i = 0; i < neighbor_num; ++i) {
         const uint64_t max_value = max( lists[i], value );
         const uint64_t min_value = min( lists[i], value );
         lists[i] = min_value;
         value = max_value;
      }
      return __uint_as_float( lists[neighbor_num - 1] >> 32 );
   }

   __device__
   void findNearestNeighbors(
      uint64_t* lists,
      const KdtreeNode* root,
      const node_type* coordinates,
      const node_type* query,
      int node_index,
      int neighbor_num,
      int size,
      int dim
   )
   {
      int depth = 0;
      int prev = -1;
      int curr = node_index;
      float max_squared_distance = INFINITY;
      while (curr >= 0) {
         const KdtreeNode* node = &root[curr];
         const int parent = node->ParentIndex;
         if (curr >= size) {
            prev = curr;
            curr = parent;
            continue;
         }

         const bool from_child = prev >= 0 && (prev == node->LeftChildIndex || prev == node->RightChildIndex);
         if (!from_child) {
            float squared_distance = 0.0f;
            for (int d = 0; d < dim; ++d) {
               const float x = query[d] - coordinates[node->Index * dim + d];
               squared_distance += x * x;
            }
            if (squared_distance <= max_squared_distance) {
               max_squared_distance = push( lists, curr, neighbor_num, squared_distance );
            }
         }

         const int axis = depth % dim;
         const node_type t = query[axis] - coordinates[node->Index * dim + axis];
         const bool right_priority = t > 0;
         const int far_child = right_priority ? node->LeftChildIndex : node->RightChildIndex;
         const int close_child = right_priority ? node->RightChildIndex : node->LeftChildIndex;

         int next = -1;
         if (prev >= 0 && prev == close_child) {
            if (far_child >= 0 && (t == 0 || t * t <= max_squared_distance)) {
               next = far_child;
               depth++;
            }
            else {
               next = parent;
               depth--;
            }
         }
         else if (prev >= 0 && prev == far_child) {
            next = parent;
            depth--;
         }
         else if (prev < 0 || prev == parent) {
            if (close_child < 0 && far_child < 0) {
               next = parent;
               depth--;
            }
            else if (close_child < 0) {
               next = far_child;
               depth++;
            }
            else {
               next = close_child;
               depth++;
            }
         }

         prev = curr;
         curr = next;
      }
   }

   __global__
   void cuFindNearestNeighbors(
      uint64_t* lists,
      const KdtreeNode* root,
      const node_type* coordinates,
      const node_type* queries,
      int node_index,
      int query_num,
      int neighbor_num,
      int size,
      int dim
   )
   {
      const auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      const auto step = static_cast<int>(blockDim.x * gridDim.x);
      for (int i = index; i < query_num; i += step) {
         const node_type* query = queries + i * dim;
         uint64_t* founds = lists + i * neighbor_num;
         findNearestNeighbors( founds, root, coordinates, query, node_index, neighbor_num, size, dim );
      }
   }

   void KdtreeCUDA::findNearestNeighbors(
      std::vector<std::vector<std::pair<float, int>>>& founds,
      const node_type* queries,
      int query_num,
      int neighbor_num
   ) const
   {
      if (Device.RootNode < 0 || Coordinates == nullptr) return;

      uint64_t* lists = nullptr;
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&lists), sizeof( uint64_t ) * neighbor_num * query_num ) );

      node_type* device_queries = nullptr;
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&device_queries), sizeof( node_type ) * query_num * Dim ) );
      CHECK_CUDA(
         hipMemcpyAsync(
            device_queries, queries, sizeof( node_type ) * query_num * Dim,
            hipMemcpyHostToDevice, Device.Stream
         )
      );

      const int block_num = divideUp( query_num, WarpSize );
      cuInitializeKNN<<<block_num, WarpSize, 0, Device.Stream>>>( lists, query_num, neighbor_num );
      CHECK_KERNEL;

      cuFindNearestNeighbors<<<block_num, WarpSize, 0, Device.Stream>>>(
         lists, Device.Root, Device.CoordinatesDevicePtr, device_queries,
         Device.RootNode, query_num, neighbor_num, Device.TupleNum, Dim
      );
      CHECK_KERNEL;

      founds.clear();
      founds.resize( query_num );
      std::vector<uint64_t> neighbors(neighbor_num);
      for (int i = 0; i < query_num; ++i) {
         CHECK_CUDA(
            hipMemcpyAsync(
               neighbors.data(), lists + i * neighbor_num, sizeof( uint64_t ) * neighbor_num,
               hipMemcpyDeviceToHost, Device.Stream
            )
         );

         for (int j = 0; j < neighbor_num; ++j) {
            const auto index = static_cast<int>(neighbors[j] & 0xffffffff);
            if (index < 0) break;

            const auto float_part = static_cast<int>(neighbors[j] >> 32);
            const float squared_distance = *reinterpret_cast<const float*>(&float_part);
            founds[i].emplace_back( squared_distance, index );
         }
      }

      CHECK_CUDA( hipFree( lists ) );
      CHECK_CUDA( hipFree( device_queries ) );
   }
}
#endif